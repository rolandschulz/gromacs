#include "hip/hip_runtime.h"
#include "stdlib.h"

#include "smalloc.h"

#include "types/simple.h" 
#include "types/nblist_box.h"
#include "cutypedefs.h"
#include "cudautils.h"

#include "gpu_nb.h"
#include "gpu_data.h"

#define CELL_SIZE           (GPU_NS_CELL_SIZE)
#define NB_DEFAULT_THREADS  (CELL_SIZE * CELL_SIZE)


/* texture reference bound to the cudata.coulomb_tab array */
texture<float, 1, hipReadModeElementType> tex_coulomb_tab;

/* source: OpenMM */
static __device__ float interpolate_coulomb_force_r(float r, float scale)
{  
    float   normalized = scale * r;
    int     index = (int) normalized;
    float   fract2 = normalized - index;
    float   fract1 = 1.0f - fract2;

    return  fract1 * tex1Dfetch(tex_coulomb_tab, index) 
            + fract2 * tex1Dfetch(tex_coulomb_tab, index + 1);
}

#include "gpu_nb_kernels.h"

/* based on the number of work units, return the number of blocks to be used 
   for the nonbonded GPU kernel */
inline int calc_nb_blocknr(int nwork_units)
{
    int retval = (nwork_units <= GRID_MAX_DIM ? nwork_units : GRID_MAX_DIM);
    if (retval != nwork_units)
    {
        gmx_fatal(FARGS, "Watch out, the number of nonbonded work units exceeds the maximum grid size (%d > %d)!",
                nwork_units, GRID_MAX_DIM);
    }
    return retval;
}

/*  Launch asynchronously the nonbonded force calculations. 

    This consists of the following (async) steps launched in the default stream 0: 
   - initilize to zero force output
   - upload x and q
   - upload shift vector
   - launch kernel
   - download forces
    
    Timing is done using the start_nb and stop_nb events.
 */
void cu_stream_nb(t_cudata d_data, 
                  const gmx_nb_atomdata_t *nbatom,
                  gmx_bool sync)
{
    int     shmem; 
    int     nb_blocks = calc_nb_blocknr(d_data->nci);
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 
    
    static gmx_bool doKernel2 = (getenv("GMX_NB_K2") != NULL);        

    /* size of force buffers in shmem */
    if (!doKernel2)
    {
        shmem =  (1 + NSUBCELL) * CELL_SIZE * CELL_SIZE * 3 * sizeof(float);
    }
    else 
    {
        shmem =  CELL_SIZE * CELL_SIZE * 3 * sizeof(float);
    }

    if (debug)
    {
        fprintf(debug, "GPU launch configuration:\n\tThread block: %dx%dx%d\n\tGrid: %dx%d\n\t#Cells/Subcells: %d/%d (%d)\n",         
        dim_block.x, dim_block.y, dim_block.z, dim_grid.x, dim_grid.y, d_data->nsi, 
        NSUBCELL, d_data->naps);
    }

    hipEventRecord(d_data->start_nb, 0);
    
    /* set the forces to 0 */
    hipMemsetAsync(d_data->f, 0, d_data->natoms * sizeof(*d_data->f), 0);

    /* upload x, q */    
    upload_cudata_async(d_data->xq, nbatom->x, d_data->natoms * sizeof(*d_data->xq), 0);

    /* upload shift vec */
    upload_cudata_async(d_data->shift_vec, nbatom->shift_vec, SHIFTS * sizeof(*d_data->shift_vec), 0);   

    /* async nonbonded calculations */        
    if (!doKernel2)
    {
        k_calc_nb_1 <<<dim_grid, dim_block, shmem, 0>>>(d_data->ci,
                                                        d_data->sj,
                                                        d_data->si,
                                                        d_data->atom_types, 
                                                        d_data->ntypes, 
                                                        d_data->xq, 
                                                        d_data->nbfp,
                                                        d_data->shift_vec,
                                                        d_data->ewald_beta,
                                                        d_data->cutoff_sq,
                                                        d_data->coulomb_tab_scale,
                                                        d_data->f);
    }
    else
    {
        k_calc_nb_2 <<<dim_grid, dim_block, shmem, 0>>>(d_data->ci,
                                                        d_data->sj,
                                                        d_data->si,
                                                        d_data->atom_types,
                                                        d_data->ntypes,
                                                        d_data->xq,
                                                        d_data->nbfp,
                                                        d_data->shift_vec,
                                                        d_data->ewald_beta,
                                                        d_data->cutoff_sq,
                                                        d_data->coulomb_tab_scale,
                                                        d_data->f);
    }
   
    if (sync)
    {
        CU_LAUNCH_ERR_SYNC("k_calc_nb");
    }
    else
    {
        CU_LAUNCH_ERR("k_calc_nb");
    }
   
    /* async copy DtoH f */
    download_cudata_async(nbatom->f, d_data->f, d_data->natoms*sizeof(*d_data->f), 0);
    hipEventRecord(d_data->stop_nb, 0);
}

/* Blocking wait for the asynchrounously launched nonbonded calculations to finish. */
void cu_blockwait_nb(t_cudata d_data, float *time)
{    
    cu_blockwait_event(d_data->stop_nb, d_data->start_nb, time);
}

/* Blocking wait for the asynchrounously launched nonbonded calculations to finish. */
void cu_blockwait_nb_OLD(t_cudata d_data, float *time)
{    
    hipError_t stat;

    // stat = hipStreamSynchronize(d_data->nb_stream);    
    stat = hipEventSynchronize(d_data->stop_nb);
    CU_RET_ERR(stat, "the async execution of nonbonded calculations has failed"); 

    stat = hipEventElapsedTime(time, d_data->start_nb, d_data->stop_nb);
    CU_RET_ERR(stat, "hipEventElapsedTime on start_nb and stop_nb failed");
}

/* Check if the nonbonded calculation has finished. */
gmx_bool cu_checkstat_nb(t_cudata d_data, float *time)
{
    hipError_t stat; 
    
    time = NULL;
    stat = hipEventQuery(d_data->stop_nb);

    /* we're done, let's calculate times*/
    if (stat == hipSuccess)
    {
        stat = hipEventElapsedTime(time, d_data->start_nb, d_data->stop_nb);
        CU_RET_ERR(stat, "hipEventElapsedTime on start_nb and stop_nb failed");
    }
    else 
    {
        /* do we have an error? */
        if (stat != hipErrorNotReady) 
        {
            CU_RET_ERR(stat, "the execution of the nonbonded calculations has failed");
        }
    }
    
    return (stat == hipSuccess ? TRUE : FALSE);
}


/* XXX:  remove, not used anyomore! */
void cu_do_nb(t_cudata d_data, rvec shift_vec[]) 
{
#if 0
    int     nb_blocks = calc_nb_blocknr(d_data->nci);
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 
    int     shmem = (1 + NSUBCELL) * CELL_SIZE * CELL_SIZE * sizeof(float4); /* force buffer */

    if (debug)
    {
        printf("~> Thread block: %dx%dx%d\n~> Grid: %dx%d\n~> #SubCell pairs: %d (%d)\n", 
            dim_block.x, dim_block.y, dim_block.z, dim_grid.x, dim_grid.y, d_data->nsi, 
            d_data->naps);
    }

    /* set the forces to 0 */
    hipMemset(d_data->f, 0, d_data->natoms*sizeof(*d_data->f));

    /* upload shift vec */
    upload_cudata(d_data->shift_vec, shift_vec, SHIFTS*sizeof(*d_data->shift_vec));   

    /* sync nonbonded calculations */      
    k_calc_nb_1<<<dim_grid, dim_block, shmem>>>(d_data->ci,
                                                  d_data->sj, 
                                                  d_data->si,
                                                  d_data->atom_types, 
                                                  d_data->ntypes, 
                                                  d_data->xq, 
                                                  d_data->nbfp,
                                                  d_data->shift_vec,
                                                  d_data->ewald_beta,
                                                  d_data->f);
    CU_LAUNCH_ERR_SYNC("k_calc_nb");
#endif 
}
