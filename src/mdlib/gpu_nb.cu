#include "hip/hip_runtime.h"
#include "stdlib.h"

#include "smalloc.h"

#include "types/simple.h" 
#include "types/nblist_box.h"
#include "cutypedefs.h"
#include "cudautils.h"

#include "gpu_nb.h"
#include "gpu_data.h"

#define CELL_SIZE           (GPU_NS_CELL_SIZE)
#define NB_DEFAULT_THREADS  (CELL_SIZE * CELL_SIZE)
#define GPU_FACEL           (138.935485)

texture<float, 1, hipReadModeElementType> texnbfp;
// __device__ __constant__ c_nbfp;

#include "gpu_nb_kernels.h"

__global__ void __empty_kernel() {}

inline int calc_nb_blocknr(int nwork_units)
{
    int retval = (nwork_units <= GRID_MAX_DIM ? nwork_units : GRID_MAX_DIM);
    if (retval != nwork_units)
    {
        gmx_warning("Watch out, the number of nonbonded work units exceeds the maximum grid size (%d > %d)!",
                nwork_units, GRID_MAX_DIM);
    }
    return retval;
}

void cu_do_nb(t_cudata d_data,rvec shiftvec[]) 
{
    int     nb_blocks = calc_nb_blocknr(d_data->nci)/d_data->cell_pair_group;
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 
    int     shmem = (1 + NSUBCELL) * CELL_SIZE * CELL_SIZE * sizeof(float4); /* force buffer */

    if (debug)
    {
        printf("~> Thread block: %dx%dx%d\n~> Grid: %dx%d\n~> #SubCell pairs: %d (%d)\n", 
            dim_block.x, dim_block.y, dim_block.z, dim_grid.x, dim_grid.y, d_data->nsi, 
            d_data->naps);
    }

    /* set the forces to 0 */
    hipMemset(d_data->f, 0, d_data->natoms*sizeof(*d_data->f));

    /* upload shift vec */
    upload_cudata(d_data->shiftvec, shiftvec, SHIFTS*sizeof(*d_data->shiftvec));   

    /* sync nonbonded calculations */   
    k_calc_nb<<<dim_grid, dim_block, shmem>>>(d_data->ci,
                                                  d_data->sj, 
                                                  d_data->si,
                                                  d_data->atom_types, 
                                                  d_data->ntypes, 
                                                  d_data->xq, 
                                                  d_data->nbfp,
                                                  d_data->shiftvec,
                                                  d_data->f);
    CU_LAUNCH_ERR_SYNC("k_calc_nb");
}

void cu_stream_nb(t_cudata d_data, 
                  /*const gmx_nblist_t *nblist, */
                  const gmx_nb_atomdata_t *nbatom,
                  rvec shiftvec[])
{
    int     nb_blocks = calc_nb_blocknr(d_data->nci)/d_data->cell_pair_group;
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 
    int     shmem =  (1 + NSUBCELL) * CELL_SIZE * CELL_SIZE * sizeof(float4); /* force buffer 4*4*CELL_SIZE^2 */
    // hipStream_t st = d_data->nb_stream;
    static int     cacheConf = 0;


    /* XXX XXX */
    if (cacheConf == 0)
    {
        printf("~> Thread block: %dx%dx%d\n~> Grid: %dx%d\n~> #Cells/Subcells: %d/%d (%d)\n",         
        dim_block.x, dim_block.y, dim_block.z, dim_grid.x, dim_grid.y, d_data->nsi, 
        NSUBCELL, d_data->naps);

        printf("cell_pair_group=%d\n", d_data->cell_pair_group);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(&k_calc_nb), hipFuncCachePreferShared); 
        cacheConf++;
    }

    hipEventRecord(d_data->start_nb, 0);
    
    /* set the forces to 0 */
    hipMemsetAsync(d_data->f, 0, d_data->natoms*sizeof(*d_data->f), 0);
    /* upload x, Q */
    upload_cudata_async(d_data->xq, nbatom->x, d_data->natoms*sizeof(*d_data->xq), 0);
    /* upload shift vec */
    upload_cudata_async(d_data->shiftvec, shiftvec, SHIFTS*sizeof(*d_data->shiftvec), 0);   

    /* async nonbonded calculations */        
#if 0
    k_calc_nb<<<dim_grid, dim_block, shmem, 0>>>(*d_data);
#else
    k_calc_nb<<<dim_grid, dim_block, shmem, 0>>>(d_data->ci,             
                                                  d_data->sj, 
                                                  d_data->si,
                                                  d_data->atom_types, 
                                                  d_data->ntypes, 
                                                  d_data->xq, 
                                                  d_data->nbfp,
                                                  d_data->shiftvec,
                                                  d_data->f);    
#endif
    CU_LAUNCH_ERR("k_calc_nb");
   
    /* async copy DtoH f */    
    download_cudata_async(nbatom->f, d_data->f, d_data->natoms*sizeof(*d_data->f), 0);
    hipEventRecord(d_data->stop_nb, 0);
}

gmx_bool cu_checkstat_nb(t_cudata d_data, float *time)
{
    hipError_t stat; 
    
    time = NULL;
    stat = hipEventQuery(d_data->stop_nb);

    /* we're done, let's calculate times*/
    if (stat == hipSuccess)
    {
        hipEventElapsedTime(time, d_data->start_nb, d_data->stop_nb);
    }
    else 
    {
        /* do we have an error? */
        if (stat != hipErrorNotReady) 
        {
            CU_RET_ERR(stat, "the execution of the nonbonded calculations has failed");
        }
    }
    
    return (stat == hipSuccess ? TRUE : FALSE);
}

void cu_blockwait_nb(t_cudata d_data, float *time)
{    
    hipError_t stat;     

    // stat = hipStreamSynchronize(d_data->nb_stream);
    stat = hipEventSynchronize(d_data->stop_nb);
    CU_RET_ERR(stat, "the async execution of nonbonded calculations has failed");   
   
    hipEventElapsedTime(time, d_data->start_nb, d_data->stop_nb);
}
