#include "hip/hip_runtime.h"
#include "stdlib.h"

#include "smalloc.h"

#include "types/simple.h" 
#include "types/nblist_box.h"
#include "cutypedefs.h"
#include "cudautils.h"

#include "gpu_nb.h"
#include "gpu_data.h"

#define CELL_SIZE           (GPU_NS_CELL_SIZE)
#define NB_DEFAULT_THREADS  (CELL_SIZE * CELL_SIZE)
#define GPU_FACEL           (138.935485)

#include "gpu_nb_kernels.h"

__global__ void __empty_kernel() {}

inline int calc_nb_blocknr(int nwork_units)
{
    int retval = (nwork_units <= GRID_MAX_DIM ? nwork_units : GRID_MAX_DIM);
    if (retval != nwork_units)
    {
        gmx_fatal(FARGS, "Watch out, the number of nonbonded work units exceeds the maximum grid size (%d > %d)!",
                nwork_units, GRID_MAX_DIM);
    }
    return retval;
}

void cu_stream_nb(t_cudata d_data, 
                  /*const gmx_nblist_t *nblist, */
                  const gmx_nb_atomdata_t *nbatom,
                  rvec shiftvec[],
                  gmx_bool sync)
{
    int     nb_blocks = calc_nb_blocknr(d_data->nci)/d_data->cell_pair_group;
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 
    /* force buffers in shmem */
    int     shmem =  (1 + NSUBCELL) * CELL_SIZE * CELL_SIZE * 3 * sizeof(float);
    // hipStream_t st = d_data->nb_stream;
    static gmx_bool  cache_conf_set = FALSE;

    /* XXX fix this cause it's ugly */
    if (!cache_conf_set)
    {
        printf("~> Thread block: %dx%dx%d\n~> Grid: %dx%d\n~> #Cells/Subcells: %d/%d (%d)\n",         
        dim_block.x, dim_block.y, dim_block.z, dim_grid.x, dim_grid.y, d_data->nsi, 
        NSUBCELL, d_data->naps);

        // printf("cell_pair_group=%d\n", d_data->cell_pair_group);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(&k_calc_nb_1), hipFuncCachePreferShared);        
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(&k_calc_nb_2), hipFuncCachePreferL1); 
        cache_conf_set = TRUE;
    }

    hipEventRecord(d_data->start_nb, 0);
    
    /* set the forces to 0 */
    hipMemsetAsync(d_data->f, 0, d_data->natoms*sizeof(*d_data->f), 0);

#if 0 // WC malloc stuff
    //**/ hipEventRecord(d_data->start_x_trans, 0); 
    ///**/ upload_cudata_async(d_data->xq, d_data->h_xq, d_data->natoms*sizeof(*d_data->xq), 0);
    ///**/ hipEventRecord(d_data->stop_x_trans, 0); 
#endif

    /* upload x, Q */    
    upload_cudata_async(d_data->xq, nbatom->x, d_data->natoms*sizeof(*d_data->xq), 0);

    /* upload shift vec */
    upload_cudata_async(d_data->shiftvec, shiftvec, SHIFTS*sizeof(*d_data->shiftvec), 0);   

    /* async nonbonded calculations */        
    k_calc_nb_1<<<dim_grid, dim_block, shmem, 0>>>(d_data->ci,             
                                                  d_data->sj, 
                                                  d_data->si,
                                                  d_data->atom_types, 
                                                  d_data->ntypes, 
                                                  d_data->xq, 
                                                  d_data->nbfp,
                                                  d_data->shiftvec,
                                                  d_data->f);    
    if (sync)
    {
        CU_LAUNCH_ERR_SYNC("k_calc_nb");
    }
    else
    {
        CU_LAUNCH_ERR("k_calc_nb");
    }
   
    /* async copy DtoH f */    
    download_cudata_async(nbatom->f, d_data->f, d_data->natoms*sizeof(*d_data->f), 0);
    hipEventRecord(d_data->stop_nb, 0);
}

gmx_bool cu_checkstat_nb(t_cudata d_data, float *time)
{
    hipError_t stat; 
    
    time = NULL;
    stat = hipEventQuery(d_data->stop_nb);

    /* we're done, let's calculate times*/
    if (stat == hipSuccess)
    {
        hipEventElapsedTime(time, d_data->start_nb, d_data->stop_nb);
    }
    else 
    {
        /* do we have an error? */
        if (stat != hipErrorNotReady) 
        {
            CU_RET_ERR(stat, "the execution of the nonbonded calculations has failed");
        }
    }
    
    return (stat == hipSuccess ? TRUE : FALSE);
}

void cu_blockwait_nb(t_cudata d_data, float *time)
{    
    hipError_t stat;

    // stat = hipStreamSynchronize(d_data->nb_stream);    
    stat = hipEventSynchronize(d_data->stop_nb);
    CU_RET_ERR(stat, "the async execution of nonbonded calculations has failed"); 
    hipEventElapsedTime(time, d_data->start_nb, d_data->stop_nb);
 
#if 0 // WC malloc stuff  
    float t;
    int static step = 0;
    stat = hipEventSynchronize(d_data->stop_x_trans);
    hipEventElapsedTime(&t, d_data->start_x_trans, d_data->stop_x_trans);
    d_data->x_trans_time += t;
    step++;
    if (step % 1000 == 0)
    {
        printf("xq transfer time (step %d): %5.3f ms\n", step, d_data->x_trans_time/step);
    }
#endif
}

/* XXX: not called anyomore! */
void cu_do_nb(t_cudata d_data, rvec shiftvec[]) 
{
    int     nb_blocks = calc_nb_blocknr(d_data->nci)/d_data->cell_pair_group;
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 
    int     shmem = (1 + NSUBCELL) * CELL_SIZE * CELL_SIZE * sizeof(float4); /* force buffer */

    if (debug)
    {
        printf("~> Thread block: %dx%dx%d\n~> Grid: %dx%d\n~> #SubCell pairs: %d (%d)\n", 
            dim_block.x, dim_block.y, dim_block.z, dim_grid.x, dim_grid.y, d_data->nsi, 
            d_data->naps);
    }

    /* set the forces to 0 */
    hipMemset(d_data->f, 0, d_data->natoms*sizeof(*d_data->f));

    /* upload shift vec */
    upload_cudata(d_data->shiftvec, shiftvec, SHIFTS*sizeof(*d_data->shiftvec));   

    /* sync nonbonded calculations */   
    k_calc_nb_1<<<dim_grid, dim_block, shmem>>>(d_data->ci,
                                                  d_data->sj, 
                                                  d_data->si,
                                                  d_data->atom_types, 
                                                  d_data->ntypes, 
                                                  d_data->xq, 
                                                  d_data->nbfp,
                                                  d_data->shiftvec,
                                                  d_data->f);
    CU_LAUNCH_ERR_SYNC("k_calc_nb");
}


