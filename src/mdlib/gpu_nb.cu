#include "hip/hip_runtime.h"
#include "stdlib.h"

#include "smalloc.h"

#include "types/simple.h" 
#include "types/nblist_box.h"
#include "cutypedefs.h"
#include "cudautils.h"

#include "gpu_nb.h"
#include "gpu_data.h"

#define CELL_SIZE           32
#define NB_DEFAULT_THREADS  (CELL_SIZE * CELL_SIZE)// 256

#include "gpu_nb_kernels.h"

__global__ void __empty_kernel() {}

inline int calc_nb_blocknr(int nwork_units)
{
    /*
    return (nwork_units % NB_DEFAULT_THREADS == 0 ? 
                nwork_units/NB_DEFAULT_THREADS : 
                nwork_units/NB_DEFAULT_THREADS + 1);
    */
    return nwork_units;
}

void cu_do_nb(t_cudata d_data)
{
    int     nb_blocks = calc_nb_blocknr(d_data->nlist);
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 
    int     shmem = CELL_SIZE * CELL_SIZE * sizeof(float4); /* force buffer */

    if (debug)
    {
        printf("~> Thread block: %dx%dx%d\n~> Grid: %dx%d\n~> #Cells: %d (%d)\n", 
            dim_block.x, dim_block.y, dim_block.z, dim_grid.x, dim_grid.y, d_data->ncj, d_data->napc);
        printf(">> executing nb kernel\n");
    }
    /* sync nonbonded calculations */
    k_calc_nb<<<dim_grid, dim_block, shmem>>>(*d_data);
    CU_SYNC_LAUNCH_ERR("k_calc_nb");
}

void cu_stream_nb(t_cudata d_data, 
                  /*const gmx_nblist_t *nblist, */
                  const gmx_nb_atomdata_t *nbatom)
{
    int     nb_blocks = calc_nb_blocknr(d_data->nlist);
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 
    int     shmem = CELL_SIZE * CELL_SIZE * sizeof(float4); /* force buffer */

    /* async copy HtoD x */
    hipEventRecord(d_data->start_nb, d_data->nb_stream);
    hipMemcpyAsync(d_data->xq, nbatom->x, d_data->natoms*sizeof(*d_data->xq), hipMemcpyHostToDevice, 0);    

    /* async nonbonded calculations */
    k_calc_nb<<<dim_grid, dim_block, shmem, d_data->nb_stream>>>(*d_data);
    CU_LAUNCH_ERR("k_calc_nb");
   
    /* async copy DtoH f */    
    hipMemcpyAsync(nbatom->f, d_data->f, d_data->natoms*sizeof(*d_data->f), hipMemcpyDeviceToHost, 0);
    hipEventRecord(d_data->stop_nb, d_data->nb_stream);
}

gmx_bool cu_checkstat_nb(t_cudata d_data, float *time)
{
    hipError_t stat; 
    
    time = NULL;
    stat = hipEventQuery(d_data->stop_nb);

    /* we're done, let's calculate times*/
    if (stat == hipSuccess)
    {
        hipEventElapsedTime(time, d_data->start_nb, d_data->stop_nb);
    }
    else 
    {
        /* do we have an error? */
        if (stat != hipErrorNotReady) 
        {
            CU_RET_ERR(stat, "the execution of the nonbonded calculations has failed");
        }
    }
    
    return (stat == hipSuccess ? TRUE : FALSE);
}

void cu_blockwait_nb(t_cudata d_data, float *time)
{    
    hipError_t stat;     

    stat = hipStreamSynchronize(d_data->nb_stream);
    CU_RET_ERR(stat, "the execution of the nonbonded calculations has failed");   
   
    hipEventElapsedTime(time, d_data->start_nb, d_data->stop_nb);
}
