#include <stdlib.h>
#include <stdio.h>

#include "gmx_fatal.h"
#include "smalloc.h"

#include "cutypedefs.h"
#include "cudautils.h"
#include "gpu_data.h"
#include "cupmalloc.h"

#define USE_CUDA_EVENT_BLOCKING_SYNC FALSE /* makes the CPU thread busy-wait! */

/*** CUDA MD Data operations ***/

/* forward declaration*/
void destroy_cudata_atoms(t_cudata /*d_data*/);
void destroy_cudata_ci(t_cudata /*d_data*/);
void destroy_cudata_sj(t_cudata /*d_data*/);
void destroy_cudata_si(t_cudata /*d_data*/);

void init_cudata_ff(FILE *fplog, 
                    t_cudata *dp_data,
                    const t_forcerec *fr)
{
    t_cudata            d_data = NULL;    
    hipError_t         stat;
    gmx_nb_atomdata_t   *nbat = fr->nbat;
    int                 ntypes = nbat->ntype;    
    char                *env_var;
    int                 itmp;

#if 0 /* texture business */
    hipChannelFormatDesc   cd;
    const textureReference  *texnbfp;
#endif

    int eventflags = ( USE_CUDA_EVENT_BLOCKING_SYNC ? hipEventBlockingSync: hipEventDefault );

    if (dp_data == NULL) return;
    
    snew(d_data, 1);

    d_data->ntypes  = ntypes;
    d_data->nalloc  = 0;
    
    d_data->eps_r = fr->epsilon_r;
    d_data->eps_rf = fr->epsilon_rf;   

    /* events for NB async ops */
    d_data->streamGPU = fr->streamGPU;
    if (d_data->streamGPU)
    {
        stat = hipEventCreateWithFlags(&(d_data->start_nb), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb failed");
        stat = hipEventCreateWithFlags(&(d_data->stop_nb), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb failed");
        stat = hipEventCreateWithFlags(&(d_data->start_atomdata), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_atomdata failed");
        stat = hipEventCreateWithFlags(&(d_data->stop_atomdata), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_atomdata failed");       
#if 0 // WC malloc stuff
        stat = hipEventCreateWithFlags(&(d_data->start_x_trans), eventflags);
        stat = hipEventCreateWithFlags(&(d_data->stop_x_trans), eventflags);
#endif
    }   

    stat = hipStreamCreate(&d_data->nb_stream);
    CU_RET_ERR(stat, "cudaSteamCreate on nb_stream failed");

    /* NB params */
    stat = hipMalloc((void **)&d_data->nbfp, 2*ntypes*ntypes*sizeof(*(d_data->nbfp)));
    CU_RET_ERR(stat, "hipMalloc failed on d_data->nbfp"); 
    upload_cudata(d_data->nbfp, nbat->nbfp, 2*ntypes*ntypes*sizeof(*(d_data->nbfp)));
#if 0 /* texture business */
    stat = hipGetTextureReference(&texnbfp, HIP_SYMBOL("texnbfp"));
    CU_RET_ERR(stat, "hipGetTextureReference on texnbfp failed");
    cd = hipCreateChannelDesc<float>();
    stat = hipBindTexture(NULL, texnbfp, d_data->nbfp, &cd, 2*d_data->ntypes*d_data->ntypes);
    CU_RET_ERR(stat, "hipBindTexture on texnbfp failed");
#endif
    stat = hipMalloc((void**)&d_data->shiftvec, SHIFTS*sizeof(*d_data->shiftvec));
    CU_RET_ERR(stat, "hipMalloc failed on d_data->shiftvec");

    if (fplog != NULL)
    {
        fprintf(fplog, "Initialized CUDA data structures.\n");
        
        printf("Initialized CUDA data structures.\n");
        fflush(stdout);
    }

    /* initilize to NULL all data structures that might need reallocation 
       in init_cudata_atoms */
    d_data->xq      = NULL;
    d_data->f       = NULL;
    d_data->ci      = NULL;
    d_data->sj      = NULL;
    d_data->si      = NULL;

    d_data->h_xq    = NULL;

    /* size -1 just means that it has not been initialized yet */
    d_data->natoms          = -1;
    d_data->nalloc          = -1;
    d_data->nci             = -1;
    d_data->ci_nalloc       = -1;
    d_data->nsj_1           = -1;
    d_data->sj_nalloc       = -1;
    d_data->nsi             = -1;
    d_data->si_nalloc       = -1;
    d_data->naps            = -1;

    if ((env_var = getenv("GMX_CELL_PAIR_GROUP")) != NULL)
    {
        sscanf(env_var, "%d", &itmp);
        if (itmp < 1)
        {
            gmx_fatal(FARGS, "Invalid GMX_CELL_PAIR_GROUP value (%d)!", itmp);
        }
        else
        {
            printf("CELL_PAIR_GROUP=%d\n", itmp);
            d_data->cell_pair_group = itmp;
        }
    }
    else 
    {
        d_data->cell_pair_group = GPU_CELL_PAIR_GROUP;
    }

    *dp_data = d_data;
}

/* TODO: move initilizations into a function! */
void init_cudata_atoms(t_cudata d_data, 
                       const gmx_nb_atomdata_t *atomdata, 
                       const gmx_nblist_t *nblist,
                       gmx_bool doStream)
{
    hipError_t stat;
    int         nalloc, ci_nalloc, sj_nalloc, si_nalloc;
    int         natoms  = atomdata->natoms;
    int         nci     = nblist->nci;
    int         nsj_1   = nblist->nsj + 1;
    int         nsi     = nblist->nsi;
   
    /* asynch copy all data */
    hipEventRecord(d_data->start_atomdata, 0);

    if (d_data->naps < 0)
    {
        d_data->naps = nblist->naps;
    }
    else
    {
        if (d_data->naps != nblist->naps)
        {
            gmx_fatal(FARGS, "Internal error: the #atoms per cell has changed (from %d to %d)",
                    d_data->naps, nblist->naps);
        }
    }

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven't initilzed yet, i.e d_data->natoms == -1 */
    if (natoms > d_data->nalloc)
    {
        nalloc = natoms * 1.2 + 100;
    
        /* free up first if the arrays have already been initialized */
        if (d_data->nalloc != -1)
        {
            destroy_cudata_atoms(d_data);                
        }
        
        stat = hipMalloc((void **)&d_data->f, nalloc*sizeof(*(d_data->f)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->f");                   
        stat = hipMalloc((void **)&d_data->xq, nalloc*sizeof(*(d_data->xq)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->xq");            
        stat = hipMalloc((void **)&d_data->atom_types, nalloc*sizeof(*(d_data->atom_types)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->atom_types"); 

        d_data->nalloc = nalloc;
    }
    /* XXX for the moment we just set all 8 values to the same value... 
       ATM not, we'll do that later */    
    d_data->natoms = natoms;

    if (nci > d_data->ci_nalloc) 
    {
        ci_nalloc = nci * 1.2 + 100;

        /* free up first if the arrays have already been initialized */
        if (d_data->ci_nalloc != -1)
        {
            destroy_cudata_ci(d_data);                
        }

        stat = hipMalloc((void **)&d_data->ci, ci_nalloc*sizeof(*(d_data->ci)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->ci");           

        d_data->ci_nalloc = ci_nalloc;
    }
    d_data->nci = nci;

    if (nsj_1 > d_data->nsj_1) 
    {
        sj_nalloc = nsj_1 * 1.2 + 100;

        /* free up first if the arrays have already been initialized */
        if (d_data->sj_nalloc != -1)
        {
            destroy_cudata_sj(d_data);                
        }

        stat = hipMalloc((void **)&d_data->sj, sj_nalloc*sizeof(*(d_data->sj)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->sj");    

        d_data->sj_nalloc = sj_nalloc;
    }
    d_data->nsj_1 = nsj_1;

    if (nsi > d_data->nsi)
    {
        si_nalloc = nsi * 1.2 + 100;

        /* free up first if the arrays have already been initialized */
        if (d_data->si_nalloc != -1)
        {
            destroy_cudata_si(d_data);                
        }

        stat = hipMalloc((void **)&d_data->si, si_nalloc*sizeof(*(d_data->si)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->si");    

        d_data->si_nalloc = si_nalloc;
    }
    d_data->nsi = nsi;

#if 0 // WC malloc stuff
    if (d_data->h_xq) 
    {
        pfree(d_data->h_xq);
    }
    pmalloc_wc((void**)&d_data->h_xq, d_data->nalloc*sizeof(*d_data->h_xq));
    memcpy(d_data->h_xq, atomdata->x, d_data->nalloc*sizeof(*d_data->h_xq));
#endif

    if(doStream)
    {
        upload_cudata_async(d_data->atom_types, atomdata->type, natoms*sizeof(*(d_data->atom_types)), 0);
        upload_cudata_async(d_data->ci, nblist->ci, nci*sizeof(*(d_data->ci)), 0);
        upload_cudata_async(d_data->sj, nblist->sj, nsj_1*sizeof(*(d_data->sj)), 0);
        upload_cudata_async(d_data->si, nblist->si, nsi*sizeof(*(d_data->si)), 0);       
    }
    else 
    {
        upload_cudata(d_data->atom_types, atomdata->type, natoms*sizeof(*(d_data->atom_types)));
        upload_cudata(d_data->ci, nblist->ci, nci*sizeof(*(d_data->ci)));
        upload_cudata(d_data->sj, nblist->sj, nsj_1*sizeof(*(d_data->sj)));
        upload_cudata(d_data->si, nblist->si, nsi*sizeof(*(d_data->si)));    
    }
    hipEventRecord(d_data->stop_atomdata, 0);
 
}

void cu_blockwait_atomdata(t_cudata d_data, float *time)
{    
    hipError_t stat;     

    stat = hipEventSynchronize(d_data->stop_atomdata);
    CU_RET_ERR(stat, "the async trasfer of atomdata has failed");   

    hipEventElapsedTime(time, d_data->start_atomdata, d_data->stop_atomdata);
}

void destroy_cudata(FILE *fplog, t_cudata d_data)
{
    hipError_t stat;
    const textureReference *texnbfp;

    if (d_data == NULL) return;

    if (d_data->streamGPU)
    {
        stat = hipEventDestroy(d_data->start_nb);
        CU_RET_ERR(stat, "hipEventDestroy failed on d_data->start_nb");
        stat = hipEventDestroy(d_data->stop_nb);
        CU_RET_ERR(stat, "hipEventDestroy failed on d_data->stop_nb");
        stat = hipStreamDestroy(d_data->nb_stream); 
        CU_RET_ERR(stat, "hipStreamDestroy failed on d_data->nb_stream");
    }

    stat = hipGetTextureReference(&texnbfp, HIP_SYMBOL("texnbfp"));
    CU_RET_ERR(stat, "hipGetTextureReference on texnbfp failed");
    stat = hipUnbindTexture(texnbfp);
    CU_RET_ERR(stat, "hipUnbindTexture failed on texnbfp");

    stat = hipFree(d_data->nbfp);
    CU_RET_ERR(stat, "hipFree failed on d_data->nbfp");

    destroy_cudata_atoms(d_data);

    destroy_cudata_ci(d_data);
    destroy_cudata_sj(d_data);
    destroy_cudata_si(d_data);

    stat = hipDeviceReset();
    CU_RET_ERR(stat, "hipDeviceReset failed");

    fprintf(fplog, "Cleaned up CUDA data structures.\n");
}

void destroy_cudata_atoms(t_cudata d_data)
{
    hipError_t stat;

    stat = hipFree(d_data->f);
    CU_RET_ERR(stat, "hipFree failed on d_data->f");
    stat = hipFree(d_data->xq);   
    CU_RET_ERR(stat, "hipFree failed on d_data->xq");
    stat = hipFree(d_data->atom_types);   
    CU_RET_ERR(stat, "hipFree failed on d_data->atom_types");
    d_data->natoms = -1;
    d_data->nalloc = -1;
}

void destroy_cudata_ci(t_cudata d_data)
{
    hipError_t stat;

    stat = hipFree(d_data->ci);
    CU_RET_ERR(stat, "hipFree failed on d_data->ci");
    d_data->nci = -1;
    d_data->ci_nalloc = -1;
}

void destroy_cudata_sj(t_cudata d_data)
{
    hipError_t stat;

    stat = hipFree(d_data->sj);
    CU_RET_ERR(stat, "hipFree failed on d_data->sj");
    d_data->nsj_1 = -1;
    d_data->sj_nalloc = -1;
}

void destroy_cudata_si(t_cudata d_data)
{
    hipError_t stat;

    stat = hipFree(d_data->si);
    CU_RET_ERR(stat, "hipFree failed on d_data->si");
    d_data->nsi = -1;
    d_data->si_nalloc = -1;
}

int cu_upload_X(t_cudata d_data, real *h_x) 
{
    if (debug) printf(">> uploading X\n");
    return upload_cudata(d_data->xq, h_x, d_data->natoms*sizeof(*d_data->xq));
}

int cu_download_F(real *h_f, t_cudata d_data)
{
    if (debug) printf(">> downloading F\n");
    return download_cudata(h_f, d_data->f, d_data->natoms*sizeof(*d_data->f));
}
