#include <stdlib.h>
#include <stdio.h>

#include "gmx_fatal.h"
#include "smalloc.h"

#include "cutypedefs.h"
#include "cudautils.h"
#include "gpu_data.h"

/*** CUDA MD Data operations ***/

/* forward declaration*/
void destroy_cudata_atoms(t_cudata /*d_data*/);

void init_cudata_ff(FILE *fplog, 
                    t_cudata *dp_data,
                    const t_forcerec *fr)
{
    t_cudata    d_data = NULL; 
    hipError_t stat;
    int         ntypes = fr->ntype;;

    if (dp_data == NULL) return;
    
    snew(d_data, 1);

    d_data->ntypes  = ntypes;
    d_data->nalloc  = 0;
    
    d_data->eps_r = fr->epsilon_r;
    d_data->eps_rf = fr->epsilon_rf;   

    stat = hipMalloc((void **)&d_data->nbfp, 2*ntypes*sizeof(*(d_data->nbfp)));
    CU_RET_ERR(stat, "hipMalloc failed on d_data->nbfp"); 
    upload_cudata(d_data->nbfp, fr->nbfp, 2*ntypes*sizeof(*(d_data->nbfp)));

    if (fplog != NULL)
    {
        fprintf(fplog, "Initialized CUDA data structures.\n");
        
        printf("Initialized CUDA data structures.\n");
        fflush(stdout);
    }

    /* initilize to NULL all data structures that might need reallocation 
       in init_cudata_atoms */
    d_data->x = NULL;
    d_data->f = NULL;
    /* size -1 just means that it has not been initialized yet */
    d_data->natoms = -1;
    d_data->nalloc = -1;
    *dp_data = d_data;
}

/* natoms gets the value of fr->natoms_force */
void init_cudata_atoms(t_cudata d_data,
                        const t_mdatoms *mdatoms,
                        int natoms)
{
    hipError_t stat;
    int         nalloc;

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven't initilzed yet aka d_data->natoms == -1 */
    if (natoms > d_data->nalloc)
    {
        nalloc = natoms * 1.2 + 100;
    
        /* free up first if the arrays have already been initialized */
        if (d_data->nalloc != -1)
        {
            destroy_cudata_atoms(d_data);                
        }
        
        stat = hipMalloc((void **)&d_data->f, nalloc*sizeof(*(d_data->f)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->f");           
        
        stat = hipMalloc((void **)&d_data->x, nalloc*sizeof(*(d_data->x)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->x");
            
        stat = hipMalloc((void **)&d_data->atom_types, nalloc*sizeof(*(d_data->atom_types)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->atom_types"); 

        stat = hipMalloc((void **)&d_data->charges, nalloc*sizeof(*(d_data->charges)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->charges");

        d_data->nalloc = nalloc;
    }

    upload_cudata(d_data->atom_types, mdatoms->typeA, natoms*sizeof(*(d_data->atom_types)));
    upload_cudata(d_data->charges, mdatoms->chargeA, natoms*sizeof(*(d_data->charges)));

    /* XXX for the moment we just set all 8 values to the same value... 
       ATM not, we'll do that later :) */    
    d_data->natoms = natoms;
}

void destroy_cudata(FILE *fplog, t_cudata d_data)
{
    hipError_t stat;

    if (d_data == NULL) return;

    stat = hipFree(d_data->nbfp);
    CU_RET_ERR(stat, "hipFree failed on d_data->nbfp");

    fprintf(fplog, "Cleaned up CUDA data structures.\n");
}

void destroy_cudata_atoms(t_cudata d_data)
{
    hipError_t stat;

    stat = hipFree(d_data->f);
    CU_RET_ERR(stat, "hipFree failed on d_data->f");
    stat = hipFree(d_data->x);   
    CU_RET_ERR(stat, "hipFree failed on d_data->x");
    stat = hipFree(d_data->atom_types);   
    CU_RET_ERR(stat, "hipFree failed on d_data->atom_types");
    stat = hipFree(d_data->charges);
    CU_RET_ERR(stat, "hipFree failed on d_data->charges");
}

int cu_upload_X(t_cudata d_data, rvec h_x[])
{
    return upload_cudata(d_data->x, h_x, d_data->natoms*sizeof(*d_data->x));
}

int cu_download_F(rvec h_f[], t_cudata d_data)
{
    return download_cudata(h_f, d_data->f, d_data->natoms*sizeof(*h_f));
}
