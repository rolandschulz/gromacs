#include <stdlib.h>
#include <stdio.h>

#include "gmx_fatal.h"
#include "smalloc.h"

#include "cutypedefs.h"
#include "cudautils.h"
#include "gpu_data.h"

#define USE_CUDA_ENVENT_BLOCKING_SYNC TRUE

/*** CUDA MD Data operations ***/

/* forward declaration*/
void destroy_cudata_atoms(t_cudata /*d_data*/);
void destroy_cudata_nblist(t_cudata /*d_data*/);
void destroy_cudata_cj(t_cudata /*d_data*/);

void init_cudata_ff(FILE *fplog, 
                    t_cudata *dp_data,
                    const t_forcerec *fr)
{
    t_cudata            d_data = NULL;    
    hipError_t         stat;
    gmx_nb_atomdata_t   *nbat = fr->nbat;
    int                 ntypes = nbat->ntype;
    
    if (dp_data == NULL) return;
    
    snew(d_data, 1);

    d_data->ntypes  = ntypes;
    d_data->nalloc  = 0;
    
    d_data->eps_r = fr->epsilon_r;
    d_data->eps_rf = fr->epsilon_rf;   

    /* events for NB async ops */
    
    if (USE_CUDA_ENVENT_BLOCKING_SYNC)
    {
        stat = hipEventCreate(&(d_data->start_nb));
    }
    else 
    {
        stat = hipEventCreateWithFlags(&(d_data->start_nb), hipEventBlockingSync);
    }
    CU_RET_ERR(stat, "hipEventCreate on start_nb failed");
    if (USE_CUDA_ENVENT_BLOCKING_SYNC)
    {
        stat = hipEventCreateWithFlags(&(d_data->stop_nb), hipEventBlockingSync);
    }
    else 
    {
        stat = hipEventCreate(&(d_data->stop_nb));       
    }
    CU_RET_ERR(stat, "hipEventCreate on stop_nb failed");
    

    /* NB params */
    stat = hipMalloc((void **)&d_data->nbfp, 2*ntypes*sizeof(*(d_data->nbfp)));
    CU_RET_ERR(stat, "hipMalloc failed on d_data->nbfp"); 
    upload_cudata(d_data->nbfp, nbat->nbfp, 2*ntypes*sizeof(*(d_data->nbfp)));

    if (fplog != NULL)
    {
        fprintf(fplog, "Initialized CUDA data structures.\n");
        
        printf("Initialized CUDA data structures.\n");
        fflush(stdout);
    }

    /* initilize to NULL all data structures that might need reallocation 
       in init_cudata_atoms */
    d_data->xq      = NULL;
    d_data->f       = NULL;
    d_data->nblist  = NULL;
    d_data->cj      = NULL;

    /* size -1 just means that it has not been initialized yet */
    d_data->natoms          = -1;
    d_data->nalloc          = -1;
    d_data->nlist           = -1;
    d_data->nblist_nalloc   = -1;
    d_data->ncj             = -1;
    d_data->cj_nalloc       = -1;
    d_data->napc            = -1;

    *dp_data = d_data;
}

/* natoms gets the value of fr->natoms_force */
void init_cudata_atoms(t_cudata d_data, 
                       const gmx_nb_atomdata_t *atomdata, 
                       const gmx_nblist_t *nblist)
{
    hipError_t stat;
    int         nalloc, nblist_nalloc, cj_nalloc;
    int         natoms  = atomdata->natoms;
    int         nlist   = nblist->nlist;
    int         ncj     = nblist->ncj;
   
    if (d_data->napc < 0)
    {
        d_data->napc = nblist->napc;
    }
    else
    {
        if (d_data->napc != nblist->napc)
        {
            gmx_fatal(FARGS, "Internal error: the #atoms per cell has changed (from %d to %d)",
                    d_data->nblist, nblist->napc);
        }
    }

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven't initilzed yet, i.e d_data->natoms == -1 */
    if (natoms > d_data->nalloc)
    {
        nalloc = natoms * 1.2 + 100;
    
        /* free up first if the arrays have already been initialized */
        if (d_data->nalloc != -1)
        {
            destroy_cudata_atoms(d_data);                
        }
        
        stat = hipMalloc((void **)&d_data->f, nalloc*sizeof(*(d_data->f)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->f");                   
        stat = hipMalloc((void **)&d_data->xq, nalloc*sizeof(*(d_data->xq)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->xq");            
        stat = hipMalloc((void **)&d_data->atom_types, nalloc*sizeof(*(d_data->atom_types)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->atom_types"); 

        d_data->nalloc = nalloc;
    }
    upload_cudata(d_data->atom_types, atomdata->type, natoms*sizeof(*(d_data->atom_types)));
    /* XXX for the moment we just set all 8 values to the same value... 
       ATM not, we'll do that later */    
    d_data->natoms = natoms;

    if (nlist > d_data->nblist_nalloc) 
    {
        nblist_nalloc = nlist * 1.2 + 100; // FIXME

        /* free up first if the arrays have already been initialized */
        if (d_data->nblist_nalloc != -1)
        {
            destroy_cudata_nblist(d_data);                
        }

        stat = hipMalloc((void **)&d_data->nblist, nblist_nalloc*sizeof(*(d_data->nblist)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->nblist");           

        d_data->nblist_nalloc = nblist_nalloc;
    }
    upload_cudata(d_data->nblist, nblist->list, nlist*sizeof(*(d_data->nblist)));
    d_data->nlist = nlist;

    if (ncj > d_data->ncj) 
    {
        cj_nalloc = ncj * 1.2 + 100; // FIXME

        /* free up first if the arrays have already been initialized */
        if (d_data->cj_nalloc != -1)
        {
            destroy_cudata_cj(d_data);                
        }

        stat = hipMalloc((void **)&d_data->cj, cj_nalloc*sizeof(*(d_data->cj)));
        CU_RET_ERR(stat, "hipMalloc failed on d_data->nblist");    

        d_data->cj_nalloc = cj_nalloc;
    }
    upload_cudata(d_data->cj, nblist->cj, ncj*sizeof(*(d_data->cj)));
    d_data->ncj = ncj;
}


void destroy_cudata(FILE *fplog, t_cudata d_data)
{
    hipError_t stat;

    if (d_data == NULL) return;

    hipEventDestroy(d_data->start_nb);
    hipEventDestroy(d_data->stop_nb);

    stat = hipFree(d_data->nbfp);
    CU_RET_ERR(stat, "hipFree failed on d_data->nbfp");

    fprintf(fplog, "Cleaned up CUDA data structures.\n");
}

void destroy_cudata_atoms(t_cudata d_data)
{
    hipError_t stat;

    stat = hipFree(d_data->f);
    CU_RET_ERR(stat, "hipFree failed on d_data->f");
    stat = hipFree(d_data->xq);   
    CU_RET_ERR(stat, "hipFree failed on d_data->xq");
    stat = hipFree(d_data->atom_types);   
    CU_RET_ERR(stat, "hipFree failed on d_data->atom_types");
    d_data->natoms = -1;
    d_data->nalloc = -1;
}

void destroy_cudata_nblist(t_cudata d_data)
{
    hipError_t stat;

    stat = hipFree(d_data->nblist);
    CU_RET_ERR(stat, "hipFree failed on d_data->nblist");
    d_data->nlist = -1;
    d_data->nblist_nalloc = -1;
}

void destroy_cudata_cj(t_cudata d_data)
{
    hipError_t stat;

    stat = hipFree(d_data->cj);
    CU_RET_ERR(stat, "hipFree failed on d_data->cj");
    d_data->ncj = -1;
    d_data->cj_nalloc = -1;
}

int cu_upload_X(t_cudata d_data, real *h_x) 
{
    if (debug) printf(">> uploading X\n");
    return upload_cudata(d_data->xq, h_x, d_data->natoms*sizeof(*d_data->xq));
}

int cu_download_F(real *h_f, t_cudata d_data)
{
    if (debug) printf(">> downloading F\n");
    return download_cudata(h_f, d_data->f, 3*d_data->natoms*sizeof(*h_f));
}
