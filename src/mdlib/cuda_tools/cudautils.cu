#include <stdlib.h>

#include "gmx_fatal.h"
#include "smalloc.h"

#include "hip/hip_runtime.h"
#include "cudautils.h"

/*** General CUDA data operations ***/
/* TODO: create a cusmalloc module that implements similar things as smalloc */

int _download_cudata_generic(void * h_dest, void * d_src, size_t bytes, 
                             gmx_bool async = FALSE, hipStream_t stream = 0)
{
    hipError_t stat;
    
    if (h_dest == 0 || d_src == 0 || bytes <= 0)
        return -1;

    if (async)
    {
        stat = hipMemcpyAsync(h_dest, d_src, bytes, hipMemcpyDeviceToHost, stream);
        CU_RET_ERR(stat, "DtoH hipMemcpyAsync failed");

    }
    else
    {
        stat = hipMemcpy(h_dest, d_src, bytes, hipMemcpyDeviceToHost);
        CU_RET_ERR(stat, "DtoH hipMemcpy failed");
    }

    return 0;
}

int download_cudata(void * h_dest, void * d_src, size_t bytes)
{
    return _download_cudata_generic(h_dest, d_src, bytes, FALSE);
}

int download_cudata_async(void * h_dest, void * d_src, size_t bytes, hipStream_t stream = 0)
{
    return _download_cudata_generic(h_dest, d_src, bytes, TRUE, stream);
}

int download_cudata_alloc(void ** h_dest, void * d_src, size_t bytes)
{ 
    if (h_dest == 0 || d_src == 0 || bytes <= 0)
        return -1;

    smalloc(*h_dest, bytes);

    return download_cudata(*h_dest, d_src, bytes);
}


int _upload_cudata_generic(void * d_dest, void * h_src, size_t bytes, 
                                 gmx_bool async = FALSE, hipStream_t stream = 0)
{
    hipError_t stat;

    if (d_dest == 0 || h_src == 0 || bytes <= 0)
        return -1;

    if (async)
    {
        stat = hipMemcpyAsync(d_dest, h_src, bytes, hipMemcpyHostToDevice, stream);
        CU_RET_ERR(stat, "HtoD hipMemcpyAsync failed");
    }
    else
    {
        stat = hipMemcpy(d_dest, h_src, bytes, hipMemcpyHostToDevice);
        CU_RET_ERR(stat, "HtoD hipMemcpy failed");
    }

    return 0;
}

int upload_cudata(void * d_dest, void * h_src, size_t bytes)
{   
    return _upload_cudata_generic(d_dest, h_src, bytes, FALSE);
}

int upload_cudata_async(void * d_dest, void * h_src, size_t bytes, hipStream_t stream = 0)
{   
    return _upload_cudata_generic(d_dest, h_src, bytes, TRUE, stream);
}

int upload_cudata_alloc(void ** d_dest, void * h_src, size_t bytes)
{
    hipError_t stat;

    if (d_dest == 0 || h_src == 0 || bytes <= 0)
        return -1;

    stat = hipMalloc(d_dest, bytes);
    CU_RET_ERR(stat, "hipMalloc failed in upload_cudata_alloc");

    return upload_cudata(*d_dest, h_src, bytes);
}

int cu_blockwait_event(hipEvent_t stop, hipEvent_t start, float *time)
{
    hipError_t s;

    s = hipEventSynchronize(stop);
    CU_RET_ERR(s, "hipEventSynchronize failed in cu_blockwait_event");

    s = hipEventElapsedTime(time, start, stop);
    CU_RET_ERR(s, "hipEventElapsedTime failed in cu_blockwait_event");

    return 0;
}
