#include <stdlib.h>

#include "gmx_fatal.h"

#include "cudautils.h"
#include "cupmalloc.h"

/* page-locked alloc */
void pmalloc(void **h_ptr, size_t nbytes)
{
    hipError_t stat;
    char        strbuf[50]; // FIXME what's the gmx macro for default small char buffers?
    int         flag = hipHostMallocDefault; // TODO put here flag selection

    if (nbytes <= 0)
    {
        *h_ptr = NULL;
        return;
    }

    CU_CHECK_PREV_ERR();

    stat = hipHostMalloc(h_ptr, nbytes, flag);    
    sprintf(strbuf, "hipHostMalloc of size %d bytes failed", (int)nbytes);
    CU_RET_ERR(stat, strbuf);  
}

void pmalloc_wc(void **h_ptr, size_t nbytes)
{
    hipError_t stat;
    char        strbuf[50]; // FIXME what's the gmx macro for default small char buffers?
    int         flag = hipHostMallocDefault || hipHostMallocWriteCombined; // TODO put here flag selection

    if (nbytes <= 0)
    {
        *h_ptr = NULL;
        return;
    }

    CU_CHECK_PREV_ERR();

    stat = hipHostMalloc(h_ptr, nbytes, flag);    
    sprintf(strbuf, "hipHostMalloc of size %d bytes failed", (int)nbytes);
    CU_RET_ERR(stat, strbuf);  
}

/* page locked free */
void pfree(void *h_ptr) 
{
    hipError_t stat; 

    if (h_ptr == NULL)
    {        
        return;
    }

    CU_CHECK_PREV_ERR();

    stat = hipHostFree(h_ptr);
    CU_RET_ERR(stat, "hipHostFree failed");
}
