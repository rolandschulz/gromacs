#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "gmx_fatal.h"
#include "smalloc.h"
#include "force.h"

#include "cutypedefs.h"
#include "cudautils.h"
#include "cuda_data_mgmt.h"
#include "cupmalloc.h"

#define USE_CUDA_EVENT_BLOCKING_SYNC FALSE /* makes the CPU thread busy-wait! */
#define EWALD_COULOMB_FORCE_TABLE_SIZE (1536)   /* size chosen such we do not run out of texture cache */

#define MY_PI               (3.1415926535897932384626433832795)
#define TWO_OVER_SQRT_PI    (2.0/sqrt(MY_PI))
    
#define TIME_GPU_TRANSFERS 1

#define NUM_NB_KERNELS 12

static const char * const nb_k1_names[NUM_NB_KERNELS] = 
{
    "_Z21k_calc_nb_RF_forces_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z24k_calc_nb_ewald_forces_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z25k_calc_nb_cutoff_forces_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z30k_calc_nb_RF_forces_energies_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z30k_calc_nb_RF_forces_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z33k_calc_nb_ewald_forces_energies_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z33k_calc_nb_ewald_forces_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z34k_calc_nb_cutoff_forces_energies_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z34k_calc_nb_cutoff_forces_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z39k_calc_nb_RF_forces_energies_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z42k_calc_nb_ewald_forces_energies_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z43k_calc_nb_cutoff_forces_energies_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti"
};

static const char * const nb_k2_names[NUM_NB_KERNELS] = 
{
    "_Z21k_calc_nb_RF_forces_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z24k_calc_nb_ewald_forces_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z25k_calc_nb_cutoff_forces_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z30k_calc_nb_RF_forces_energies_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z30k_calc_nb_RF_forces_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z33k_calc_nb_ewald_forces_energies_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z33k_calc_nb_ewald_forces_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z34k_calc_nb_cutoff_forces_energies_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z34k_calc_nb_cutoff_forces_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z39k_calc_nb_RF_forces_energies_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z42k_calc_nb_ewald_forces_energies_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z43k_calc_nb_cutoff_forces_energies_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti"
};

__device__ __global__ void k_empty_test(){}

/*** CUDA Data operations ***/
static void destroy_cudata_array(void * d_ptr, 
                                 int * n = NULL, int * nalloc = NULL);
static void realloc_cudata_array(void **d_dest, void *h_src, size_t type_size, 
                                 int *curr_size, int *curr_alloc_size, 
                                 int req_size, gmx_bool doStream);                                
static void init_ewald_coulomb_force_table(cu_nb_params_t *nb_params);

/*! Tabulates the Ewald Coulomb force and initializes the related GPU resources. 
 */
static void init_ewald_coulomb_force_table(cu_nb_params_t *nb_params)
{
    float       *ftmp, *coul_tab;
    int         tabsize;
    double      tabscale;
    hipError_t stat;

    tabsize     = EWALD_COULOMB_FORCE_TABLE_SIZE;
    tabscale    = (tabsize - 1) / sqrt(nb_params->cutoff_sq);

    pmalloc((void**)&ftmp, tabsize*sizeof(*ftmp));

    table_spline3_fill_ewald_force(ftmp, tabsize, 1/tabscale, nb_params->ewald_beta);

    stat = hipMalloc((void **)&coul_tab, tabsize*sizeof(*coul_tab));
    CU_RET_ERR(stat, "hipMalloc failed on coul_tab");
    upload_cudata(coul_tab, ftmp, tabsize*sizeof(*coul_tab));
    cu_bind_texture("tex_coulomb_tab", coul_tab, tabsize*sizeof(*coul_tab));

    nb_params->coulomb_tab          = coul_tab;
    nb_params->coulomb_tab_size     = tabsize;
    nb_params->coulomb_tab_scale    = tabscale;

    pfree(ftmp);
}


/*! Initilizes the atomdata (XXX) data structure. */
void init_atomdata(cu_atomdata_t *ad, int ntypes)
{
    hipError_t stat;

    ad->ntypes  = ntypes;
    stat = hipMalloc((void**)&ad->shift_vec, SHIFTS*sizeof(*ad->shift_vec));
    CU_RET_ERR(stat, "hipMalloc failed on ad->shift_vec"); 
    ad->shift_vec_copied = FALSE;

    stat = hipMalloc((void**)&ad->f_shift, SHIFTS*sizeof(*ad->f_shift));
    CU_RET_ERR(stat, "hipMalloc failed on ad->f_shift");

    stat = hipMalloc((void**)&ad->e_lj, sizeof(*ad->e_lj));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_lj");
    stat = hipMalloc((void**)&ad->e_el, sizeof(*ad->e_el));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_el");

    /* initilize to NULL poiters to data that is not allocated here and will
       need reallocation in init_cudata_atoms */
    ad->xq = NULL;
    ad->f  = NULL;

    /* size -1 indicates that the repective array hasn't been initialized yet */
    ad->natoms = -1;
    ad->nalloc = -1;
}

/*! Initilizes the nonbonded parameter data structure. */
void init_nb_params(cu_nb_params_t *nbp, const t_forcerec *fr)
{  
    hipError_t stat;
    int         ntypes, nnbfp; 

    ntypes = fr->nbat->ntype;
    
    nbp->ewald_beta  = fr->ewaldcoeff;
    nbp->eps_r       = fr->epsilon_r;
    nbp->two_k_rf    = 2.0 * fr->k_rf;
    nbp->c_rf        = fr->c_rf;
    nbp->cutoff_sq   = fr->rvdw * fr->rvdw;
    nbp->rlist_sq    = fr->rlist * fr->rlist;
    nbp->lj_shift    = (getenv("GMX_LJ_SHIFT") == NULL) ?
             0.0 : -1/(nbp->cutoff_sq * nbp->cutoff_sq * nbp->cutoff_sq);

    if (fr->eeltype == eelCUT)
    {
        nbp->eeltype = cu_eelCUT;
    }
    else if (EEL_RF(fr->eeltype))
    {                
        nbp->eeltype = cu_eelRF;
    }
    else if ((EEL_PME(fr->eeltype) || fr->eeltype==eelEWALD))
    {
        nbp->eeltype = cu_eelEWALD;
    }
    else 
    {
        gmx_fatal(FARGS, "The requested electrostatics type is not implemented in the CUDA GPU accelerated kernels!");
    }

    /* generate table for PME */
    if (nbp->eeltype == cu_eelEWALD)
    {
        init_ewald_coulomb_force_table(nbp);
    }

    nnbfp = 2*ntypes*ntypes;
    stat = hipMalloc((void **)&nbp->nbfp, nnbfp*sizeof(*nbp->nbfp));
    CU_RET_ERR(stat, "hipMalloc failed on nbp->nbfp"); 
    upload_cudata(nbp->nbfp, fr->nbat->nbfp, nnbfp*sizeof(*nbp->nbfp));
    cu_bind_texture("tex_nbfp", nbp->nbfp, nnbfp*sizeof(*nbp->nbfp));
}

/*! Initilizes the neighborlist data structure. */
void init_nblist(cu_nblist_t *nbl)
{
    /* initilize to NULL poiters to data that is not allocated here and will
       need reallocation in init_cudata_atoms */
    nbl->ci      = NULL;
    nbl->sj4     = NULL;
    nbl->excl    = NULL;    
    
    /* size -1 indicates that the repective array hasn't been initialized yet */
    nbl->naps        = -1;
    nbl->nci         = -1;
    nbl->ci_nalloc   = -1;
    nbl->nsj4        = -1;
    nbl->sj4_nalloc  = -1;
    nbl->nexcl       = -1;
    nbl->excl_nalloc = -1;
    nbl->prune_nbl   = FALSE;
}

/*! Initilizes the timer data structure. */
static void init_timers(cu_timers_t *t)
{
    hipError_t stat;
    /* XXX */ 
    int eventflags = ( USE_CUDA_EVENT_BLOCKING_SYNC ? hipEventBlockingSync: hipEventDefault );

    stat = hipEventCreateWithFlags(&(t->start_nb), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_nb failed");
    stat = hipEventCreateWithFlags(&(t->stop_nb), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_nb failed");
    stat = hipEventCreateWithFlags(&(t->start_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_atdat failed");
    stat = hipEventCreateWithFlags(&(t->stop_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_atdat failed");

    t->time_transfers = TIME_GPU_TRANSFERS > 0; /* XXX fix this! */

    if (t->time_transfers)
    {
        stat = hipEventCreateWithFlags(&(t->start_nb_h2d), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_h2d failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_h2d), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_h2d failed");

        stat = hipEventCreateWithFlags(&(t->start_nb_d2h), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_d2h failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_d2h), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_d2h failed");
    }
}

/*! Initilizes the timer data structure. */
static void init_timings(cu_timings_t *t)
{
    int i, j;

    t->nb_h2d_time = 0.0;
    t->nb_d2h_time = 0.0;
    t->nb_count    = 0;
    t->atomdt_h2d_total_time = 0.0;
    t->atomdt_count = 0;
    for (i = 0; i < 2; i++)
    {
        for(j = 0; j < 2; j++)
        {
            t->k_time[i][j].t = 0.0;
            t->k_time[i][j].c = 0;
        }
    }
}

/*! Initilizes force-field related data (called only once in the beginning).
 */
void init_cudata_ff(FILE *fplog, 
                    cu_nonbonded_t *p_cu_nb,
                    const t_forcerec *fr)
{
    hipError_t     stat;
    cu_nonbonded_t  nb;

    if (p_cu_nb == NULL) return;
    
    snew(nb, 1); 
    snew(nb->atomdata, 1); 
    snew(nb->nb_params, 1); 
    snew(nb->nblist, 1); 
    snew(nb->timers, 1); 
    snew(nb->timings, 1); 

    init_atomdata(nb->atomdata, fr->nbat->ntype);
    init_nb_params(nb->nb_params, fr);
    init_nblist(nb->nblist);
    init_timers(nb->timers);
    init_timings(nb->timings);

    /* init tmpdata */
    pmalloc((void**)&nb->tmpdata.e_lj, sizeof(*nb->tmpdata.e_lj));
    pmalloc((void**)&nb->tmpdata.e_el, sizeof(*nb->tmpdata.e_el));
    pmalloc((void**)&nb->tmpdata.f_shift, SHIFTS * sizeof(*nb->tmpdata.f_shift));

    nb->streamGPU   = fr->streamGPU;
    *p_cu_nb = nb;

    if (fplog != NULL)
    {
        fprintf(fplog, "Initialized CUDA data structures.\n");
    }

    /* k_calc_nb_*_1 48/16 kB Shared/L1 */
    for (int i = 0; i < NUM_NB_KERNELS; i++)
    {
        stat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_k1_names[i]),  hipFuncCachePreferShared);
        // printf("--> %s\n", nb_k1_names[i]);
        CU_RET_ERR(stat, "hipFuncSetCacheConfig failed");
    }

    /* k_calc_nb_*_2 16/48 kB Shared/L1 */
    for (int i = 0; i < NUM_NB_KERNELS; i++)
    {
        stat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_k2_names[i]), hipFuncCachePreferL1);
        CU_RET_ERR(stat, "hipFuncSetCacheConfig failed");
    }

    /* TODO: move this to gpu_utils module */
    k_empty_test<<<1, 512>>>();
    CU_LAUNCH_ERR_SYNC("test kernel");
}

/*! Initilizes atom-data and neighbor list for the GPU, called at every 
 *  neighbor search step. 
 */
void init_cudata_atoms(cu_nonbonded_t cu_nb, 
                       const gmx_nb_atomdata_t *nbat, 
                       const gmx_nblist_t *h_nblist,
                       gmx_bool doStream)
{
    hipError_t stat;
    char        sbuf[200];
    int         nalloc;
    int         natoms  = nbat->natoms;
    cu_nblist_t *d_nblist   = cu_nb->nblist;
    cu_atomdata_t *d_atomd  = cu_nb->atomdata;
    cu_timers_t *timers     = cu_nb->timers;  
    
    /* time async copy */
    stat = hipEventRecord(timers->start_atdat, 0);
    CU_RET_ERR(stat, "hipEventRecord failed on timers->start_atdat");

    if (d_nblist->naps < 0)
    {
        d_nblist->naps = h_nblist->naps;
    }
    else
    {
        if (d_nblist->naps != h_nblist->naps)
        {
            sprintf(sbuf, "In init_cudata_atoms: the #atoms per cell has changed (from %d to %d)",
                    d_nblist->naps, h_nblist->naps);            
            gmx_incons(sbuf);
        }
    }

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven"t initilzed yet, i.e d_nblist->natoms == -1 */
    if (natoms > d_atomd->nalloc)
    {
        nalloc = natoms * 1.2 + 100;
    
        /* free up first if the arrays have already been initialized */
        if (d_atomd->nalloc != -1)
        {
            destroy_cudata_array(d_atomd->f, &d_atomd->natoms, &d_atomd->nalloc);
            destroy_cudata_array(d_atomd->xq);
            destroy_cudata_array(d_atomd->atom_types, &d_atomd->ntypes);             
        }
        
        stat = hipMalloc((void **)&d_atomd->f, nalloc*sizeof(*(d_atomd->f)));
        CU_RET_ERR(stat, "hipMalloc failed on d_atomd->f");                   
        stat = hipMalloc((void **)&d_atomd->xq, nalloc*sizeof(*(d_atomd->xq)));
        CU_RET_ERR(stat, "hipMalloc failed on d_atomd->xq");     

        stat = hipMalloc((void **)&d_atomd->atom_types, nalloc*sizeof(*(d_atomd->atom_types)));
        CU_RET_ERR(stat, "hipMalloc failed on d_atomd->atom_types"); 

        d_atomd->nalloc = nalloc;
    }
    
    d_atomd->natoms = natoms;

    if(doStream)
    {
        upload_cudata_async(d_atomd->atom_types, nbat->type, 
                            natoms*sizeof(*d_atomd->atom_types), 0);
    }
    else 
    {
        upload_cudata(d_atomd->atom_types, nbat->type, 
                      natoms*sizeof(*(d_atomd->atom_types)));
    
    }

    realloc_cudata_array((void **)&d_nblist->ci, h_nblist->ci, sizeof(*(d_nblist->ci)),
                         &d_nblist->nci, &d_nblist->ci_nalloc,
                         h_nblist->nci, doStream);

    realloc_cudata_array((void **)&d_nblist->sj4, h_nblist->sj4, sizeof(*(d_nblist->sj4)),
                         &d_nblist->nsj4, &d_nblist->sj4_nalloc,
                         h_nblist->nsj4, doStream);

    realloc_cudata_array((void **)&d_nblist->excl, h_nblist->excl, sizeof(*(d_nblist->excl)),
                         &d_nblist->nexcl, &d_nblist->excl_nalloc,
                         h_nblist->nexcl, doStream);

    stat = hipEventRecord(timers->stop_atdat, 0);
    CU_RET_ERR(stat, "hipEventRecord failed on timers->stop_atdat");

    cu_nb->timings->atomdt_count++;

    /* need to prune the neighbor list during the next step */
    d_nblist->prune_nbl = TRUE;
}

/*! Frees up all GPU resources used for the nonbonded calculations. */
void destroy_cudata(FILE *fplog, cu_nonbonded_t cu_nb)
{
    hipError_t     stat;
    cu_atomdata_t   *atomdata;
    cu_nb_params_t  *nb_params;
    cu_nblist_t     *nblist;
    cu_timers_t     *timers;

    atomdata    = cu_nb->atomdata;
    nb_params   = cu_nb->nb_params;
    nblist      = cu_nb->nblist;
    timers      = cu_nb->timers;

    if (cu_nb == NULL) return;

    if (nb_params->eeltype == cu_eelEWALD)
    {
        cu_unbind_texture("tex_coulomb_tab");
        destroy_cudata_array(nb_params->coulomb_tab, &nb_params->coulomb_tab_size);            
    }

    stat = hipEventDestroy(timers->start_nb);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb");
    stat = hipEventDestroy(timers->stop_nb);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb");
    stat = hipEventDestroy(timers->start_atdat);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_atdat");
    stat = hipEventDestroy(timers->stop_atdat);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_atdat");

    if (timers->time_transfers)
    {
        stat = hipEventDestroy(timers->start_nb_h2d);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_h2d");
        stat = hipEventDestroy(timers->stop_nb_h2d);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_h2d");

        stat = hipEventDestroy(timers->start_nb_d2h);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_d2h");
        stat = hipEventDestroy(timers->stop_nb_d2h);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_d2h");
    }

    cu_unbind_texture("tex_nbfp");
    destroy_cudata_array(nb_params->nbfp);

    stat = hipFree(atomdata->shift_vec);
    CU_RET_ERR(stat, "hipEventDestroy failed on atomdata->shift_vec");
    stat = hipFree(atomdata->f_shift);
    CU_RET_ERR(stat, "hipEventDestroy failed on atomdata->f_shift");

    stat = hipFree(atomdata->e_lj);
    CU_RET_ERR(stat, "hipEventDestroy failed on atomdata->e_lj");
    stat = hipFree(atomdata->e_el);
    CU_RET_ERR(stat, "hipEventDestroy failed on atomdata->e_el");

    destroy_cudata_array(atomdata->f, &atomdata->natoms, &atomdata->nalloc);
    destroy_cudata_array(atomdata->xq);
    destroy_cudata_array(atomdata->atom_types, &atomdata->ntypes);            

    destroy_cudata_array(nblist->ci, &nblist->nci, &nblist->ci_nalloc);
    destroy_cudata_array(nblist->sj4, &nblist->nsj4, &nblist->sj4_nalloc);
    destroy_cudata_array(nblist->excl, &nblist->nexcl, &nblist->excl_nalloc);

    stat = hipDeviceReset();
    CU_RET_ERR(stat, "hipDeviceReset failed");

    fprintf(fplog, "Cleaned up CUDA data structures.\n");
}

/*! Frees the device memory pointed by d_ptr and resets the associated 
 *  size and allocation size variables to -1.
 */
static void destroy_cudata_array(void *d_ptr, int *n, int *nalloc)
{
    hipError_t stat;
    
    if (d_ptr)
    {
        stat = hipFree(d_ptr);
        CU_RET_ERR(stat, "hipFree failed");
    }

    if (n)
    {        
        *n = -1;
    }

    if (nalloc)
    {
        *nalloc = -1;
    }
}

/*! Reallocates the device memory pointed by d_ptr and copies the data from the 
 * location pointed by h_src host-side pointer. Allocation is buffered and 
 * therefor freeing is only needed if the previously allocated space is not 
 * enough. 
 */
static void realloc_cudata_array(void **d_dest, void *h_src, size_t type_size, 
                                 int *curr_size, int *curr_alloc_size, 
                                 int req_size, gmx_bool doStream)
{
    hipError_t stat;

    if (d_dest == NULL || req_size <= 0)
    {
        return;
    }

    /* reallocate only if the data does not fit = allocation size is smaller 
       than the current requested size */
    if (req_size > *curr_alloc_size)
    {
        /* only free if the array has already been initialized */
        if (*curr_alloc_size >= 0)
        {
            destroy_cudata_array(*d_dest, curr_size, curr_alloc_size);
        }

        *curr_alloc_size = 1.2 * req_size + 100;  /* TODO replace this with a fn pointer 
                                                     passed from outside */

        stat = hipMalloc(d_dest, *curr_alloc_size * type_size);
        CU_RET_ERR(stat, "hipMalloc failed in realloc_cudata_array");
    }

    /* size could have changed without actual reallocation */
    *curr_size = req_size;

    /* upload to device */
    if (h_src)
    {
        if(doStream)
        {
            upload_cudata_async(*d_dest, h_src, *curr_size * type_size, 0);
        }
        else 
        {
            upload_cudata(*d_dest, h_src,  *curr_size * type_size);
        }
    }
}

/*! Blocking waits until the atom data gets copied to the GPU and times the transfer.
 */
void cu_blockwait_atomdata(cu_nonbonded_t cu_nb)
{   
    float t;
    cu_blockwait_event(cu_nb->timers->stop_atdat, cu_nb->timers->start_atdat, &t);
    cu_nb->timings->atomdt_h2d_total_time += t;
}

/*! Returns the GPU timing structure or NULL if cu_nb is NULL. */
cu_timings_t * get_gpu_timings(cu_nonbonded_t cu_nb)
{
    return cu_nb != NULL ? cu_nb->timings : NULL;
}

/*! Resets GPU timers. */
void reset_gpu_timings(cu_nonbonded_t cu_nb)
{
    init_timings(cu_nb->timings);
}

/*** Old stuff ***/
int cu_upload_X(cu_nonbonded_t cu_nb, real *h_x) 
{
    cu_atomdata_t *ad = cu_nb->atomdata;

    return upload_cudata(ad->xq, h_x, ad->natoms*sizeof(*ad->xq));
}

int cu_download_F(real *h_f, cu_nonbonded_t cu_nb)
{
    cu_atomdata_t *ad = cu_nb->atomdata;

    return download_cudata(h_f, ad->f, ad->natoms*sizeof(*ad->f));
}
