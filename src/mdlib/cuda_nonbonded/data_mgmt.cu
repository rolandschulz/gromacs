#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "gmx_fatal.h"
#include "smalloc.h"
#include "force.h"

#include "cutypedefs.h"
#include "cudautils.h"
#include "cuda_data_mgmt.h"
#include "cupmalloc.h"

#define USE_CUDA_EVENT_BLOCKING_SYNC FALSE /* makes the CPU thread busy-wait! */
#define EWALD_COULOMB_FORCE_TABLE_SIZE (1536)   /* size chosen such we do not run out of texture cache */

#define MY_PI               (3.1415926535897932384626433832795)
#define TWO_OVER_SQRT_PI    (2.0/sqrt(MY_PI))
    
#define TIME_GPU_TRANSFERS 1

#define NUM_NB_KERNELS 12

static const char * const nb_k1_names[NUM_NB_KERNELS] = 
{
    "_Z21k_calc_nb_RF_forces_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z24k_calc_nb_ewald_forces_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z25k_calc_nb_cutoff_forces_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z30k_calc_nb_RF_forces_energies_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z30k_calc_nb_RF_forces_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z33k_calc_nb_ewald_forces_energies_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z33k_calc_nb_ewald_forces_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z34k_calc_nb_cutoff_forces_energies_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z34k_calc_nb_cutoff_forces_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z39k_calc_nb_RF_forces_energies_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z42k_calc_nb_ewald_forces_energies_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z43k_calc_nb_cutoff_forces_energies_prunenbl_111cu_atomdata12cu_nb_params9cu_nblisti"
};

static const char * const nb_k2_names[NUM_NB_KERNELS] = 
{
    "_Z21k_calc_nb_RF_forces_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z24k_calc_nb_ewald_forces_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z25k_calc_nb_cutoff_forces_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z30k_calc_nb_RF_forces_energies_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z30k_calc_nb_RF_forces_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z33k_calc_nb_ewald_forces_energies_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z33k_calc_nb_ewald_forces_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z34k_calc_nb_cutoff_forces_energies_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z34k_calc_nb_cutoff_forces_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z39k_calc_nb_RF_forces_energies_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z42k_calc_nb_ewald_forces_energies_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti",
    "_Z43k_calc_nb_cutoff_forces_energies_prunenbl_211cu_atomdata12cu_nb_params9cu_nblisti"
};

__device__ __global__ void k_empty_test(){}

/*** CUDA Data operations ***/
static void destroy_cudata_array(void * d_ptr, 
                                 int * n = NULL, int * nalloc = NULL);
static void realloc_cudata_array(void **d_dest, void *h_src, size_t type_size, 
                                 int *curr_size, int *curr_alloc_size, 
                                 int req_size,
                                 hipStream_t stream, gmx_bool doStream);
static void init_ewald_coulomb_force_table(cu_nb_params_t *nb_params);

/*! Tabulates the Ewald Coulomb force and initializes the related GPU resources. 
 */
static void init_ewald_coulomb_force_table(cu_nb_params_t *nb_params)
{
    float       *ftmp, *coul_tab;
    int         tabsize;
    double      tabscale;
    hipError_t stat;

    tabsize     = EWALD_COULOMB_FORCE_TABLE_SIZE;
    tabscale    = (tabsize - 1) / sqrt(nb_params->cutoff_sq);

    pmalloc((void**)&ftmp, tabsize*sizeof(*ftmp));

    table_spline3_fill_ewald_force(ftmp, tabsize, 1/tabscale, nb_params->ewald_beta);

    stat = hipMalloc((void **)&coul_tab, tabsize*sizeof(*coul_tab));
    CU_RET_ERR(stat, "hipMalloc failed on coul_tab");
    upload_cudata(coul_tab, ftmp, tabsize*sizeof(*coul_tab));
    cu_bind_texture("tex_coulomb_tab", coul_tab, tabsize*sizeof(*coul_tab));

    nb_params->coulomb_tab          = coul_tab;
    nb_params->coulomb_tab_size     = tabsize;
    nb_params->coulomb_tab_scale    = tabscale;

    pfree(ftmp);
}


/*! Initilizes the atomdata (XXX) data structure. */
void init_atomdata(cu_atomdata_t *ad, int ntypes)
{
    hipError_t stat;

    ad->ntypes  = ntypes;
    stat = hipMalloc((void**)&ad->shift_vec, SHIFTS*sizeof(*ad->shift_vec));
    CU_RET_ERR(stat, "hipMalloc failed on ad->shift_vec"); 
    ad->shift_vec_copied = FALSE;

    stat = hipMalloc((void**)&ad->f_shift, SHIFTS*sizeof(*ad->f_shift));
    CU_RET_ERR(stat, "hipMalloc failed on ad->f_shift");

    stat = hipMalloc((void**)&ad->e_lj, sizeof(*ad->e_lj));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_lj");
    stat = hipMalloc((void**)&ad->e_el, sizeof(*ad->e_el));
    CU_RET_ERR(stat, "hipMalloc failed on ad->e_el");

    /* initilize to NULL poiters to data that is not allocated here and will
       need reallocation in init_cudata_atoms */
    ad->xq = NULL;
    ad->f  = NULL;

    /* size -1 indicates that the repective array hasn't been initialized yet */
    ad->natoms = -1;
    ad->nalloc = -1;
}

/*! Initilizes the nonbonded parameter data structure. */
void init_nb_params(cu_nb_params_t *nbp, const t_forcerec *fr)
{  
    hipError_t stat;
    int         ntypes, nnbfp; 

    ntypes = fr->nbat->ntype;
    
    nbp->ewald_beta  = fr->ewaldcoeff;
    nbp->eps_r       = fr->epsilon_r;
    nbp->two_k_rf    = 2.0 * fr->k_rf;
    nbp->c_rf        = fr->c_rf;
    nbp->cutoff_sq   = fr->rvdw * fr->rvdw;
    nbp->rlist_sq    = fr->rlist * fr->rlist;
    nbp->lj_shift    = (getenv("GMX_LJ_SHIFT") == NULL) ?
             0.0 : -1/(nbp->cutoff_sq * nbp->cutoff_sq * nbp->cutoff_sq);

    if (fr->eeltype == eelCUT)
    {
        nbp->eeltype = cu_eelCUT;
    }
    else if (EEL_RF(fr->eeltype))
    {                
        nbp->eeltype = cu_eelRF;
    }
    else if ((EEL_PME(fr->eeltype) || fr->eeltype==eelEWALD))
    {
        nbp->eeltype = cu_eelEWALD;
    }
    else 
    {
        gmx_fatal(FARGS, "The requested electrostatics type is not implemented in the CUDA GPU accelerated kernels!");
    }

    /* generate table for PME */
    if (nbp->eeltype == cu_eelEWALD)
    {
        init_ewald_coulomb_force_table(nbp);
    }

    nnbfp = 2*ntypes*ntypes;
    stat = hipMalloc((void **)&nbp->nbfp, nnbfp*sizeof(*nbp->nbfp));
    CU_RET_ERR(stat, "hipMalloc failed on nbp->nbfp"); 
    upload_cudata(nbp->nbfp, fr->nbat->nbfp, nnbfp*sizeof(*nbp->nbfp));
    cu_bind_texture("tex_nbfp", nbp->nbfp, nnbfp*sizeof(*nbp->nbfp));
}

/*! Initilizes the neighborlist data structure. */
void init_nblist(cu_nblist_t *nbl)
{
    /* initilize to NULL poiters to data that is not allocated here and will
       need reallocation in init_cudata_atoms */
    nbl->ci      = NULL;
    nbl->sj4     = NULL;
    nbl->excl    = NULL;    
    
    /* size -1 indicates that the repective array hasn't been initialized yet */
    nbl->naps        = -1;
    nbl->nci         = -1;
    nbl->ci_nalloc   = -1;
    nbl->nsj4        = -1;
    nbl->sj4_nalloc  = -1;
    nbl->nexcl       = -1;
    nbl->excl_nalloc = -1;
    nbl->prune_nbl   = FALSE;
}

/*! Initilizes the timer data structure. */
static void init_timers(cu_timers_t *t)
{
    hipError_t stat;
    /* XXX */ 
    int eventflags = ( USE_CUDA_EVENT_BLOCKING_SYNC ? hipEventBlockingSync: hipEventDefault );

    stat = hipEventCreateWithFlags(&(t->start_nb), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_nb failed");
    stat = hipEventCreateWithFlags(&(t->stop_nb), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_nb failed");
    stat = hipEventCreateWithFlags(&(t->start_nb_nl), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_nb_nl failed");
    stat = hipEventCreateWithFlags(&(t->stop_nb_nl), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_nb_nl failed");

    stat = hipEventCreateWithFlags(&(t->start_clear), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_clear failed");
    stat = hipEventCreateWithFlags(&(t->stop_clear), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_clear failed");

    stat = hipEventCreateWithFlags(&(t->start_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_atdat failed");
    stat = hipEventCreateWithFlags(&(t->stop_atdat), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_atdat failed");
    stat = hipEventCreateWithFlags(&(t->start_atdat_nl), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on start_atdat_nl failed");
    stat = hipEventCreateWithFlags(&(t->stop_atdat_nl), eventflags);
    CU_RET_ERR(stat, "hipEventCreate on stop_atdat_nl failed");

    stat = hipStreamCreate(&t->nbstream);
    CU_RET_ERR(stat, "hipStreamCreate on nbstream failed");
    stat = hipStreamCreate(&t->nbstream_nl);
    CU_RET_ERR(stat, "hipStreamCreate on nbstream_nl failed");

    t->time_transfers = TIME_GPU_TRANSFERS > 0; /* XXX fix this! */

    if (t->time_transfers)
    {
        stat = hipEventCreateWithFlags(&(t->start_nb_h2d), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_h2d failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_h2d), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_h2d failed");
        stat = hipEventCreateWithFlags(&(t->start_nb_h2d_nl), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_h2d_nl failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_h2d_nl), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_h2d_nl failed");

        stat = hipEventCreateWithFlags(&(t->start_nb_d2h), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_d2h failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_d2h), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_d2h failed");
        stat = hipEventCreateWithFlags(&(t->start_nb_d2h_nl), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on start_nb_d2h_nl failed");
        stat = hipEventCreateWithFlags(&(t->stop_nb_d2h_nl), eventflags);
        CU_RET_ERR(stat, "hipEventCreate on stop_nb_d2h_nl failed");
    }
}

/*! Initilizes the timer data structure. */
static void init_timings(cu_timings_t *t)
{
    int i, j;

    t->nb_h2d_time = 0.0;
    t->nb_d2h_time = 0.0;
    t->nb_count    = 0;
    t->atomdt_h2d_total_time = 0.0;
    t->atomdt_count = 0;
    for (i = 0; i < 2; i++)
    {
        for(j = 0; j < 2; j++)
        {
            t->k_time[i][j].t = 0.0;
            t->k_time[i][j].c = 0;
        }
    }
}

/*! Initilizes force-field related data (called only once in the beginning).
 */
void init_cudata_ff(FILE *fplog, 
                    cu_nonbonded_t *p_cu_nb,
                    const t_forcerec *fr)
{
    hipError_t     stat;
    cu_nonbonded_t  nb;

    if (p_cu_nb == NULL) return;
    
    snew(nb, 1); 
    snew(nb->atomdata, 1); 
    snew(nb->nb_params, 1); 
    snew(nb->nblist, 1); 
    snew(nb->nblist_nl, 1);
    snew(nb->timers, 1); 
    snew(nb->timings, 1); 

    init_atomdata(nb->atomdata, fr->nbat->ntype);
    init_nb_params(nb->nb_params, fr);
    init_nblist(nb->nblist);
    init_nblist(nb->nblist_nl);
    init_timers(nb->timers);
    init_timings(nb->timings);

    /* init tmpdata */
    pmalloc((void**)&nb->tmpdata.e_lj, sizeof(*nb->tmpdata.e_lj));
    pmalloc((void**)&nb->tmpdata.e_el, sizeof(*nb->tmpdata.e_el));
    pmalloc((void**)&nb->tmpdata.f_shift, SHIFTS * sizeof(*nb->tmpdata.f_shift));

    nb->streamGPU   = fr->streamGPU;
    *p_cu_nb = nb;

    if (debug)
    {
        fprintf(debug, "Initialized CUDA data structures.\n");
    }

    /* k_calc_nb_*_1 48/16 kB Shared/L1 */
    for (int i = 0; i < NUM_NB_KERNELS; i++)
    {
        stat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_k1_names[i]),  hipFuncCachePreferShared);
        // printf("--> %s\n", nb_k1_names[i]);
        CU_RET_ERR(stat, "hipFuncSetCacheConfig failed");
    }

    /* k_calc_nb_*_2 16/48 kB Shared/L1 */
    for (int i = 0; i < NUM_NB_KERNELS; i++)
    {
        stat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(nb_k2_names[i]), hipFuncCachePreferL1);
        CU_RET_ERR(stat, "hipFuncSetCacheConfig failed");
    }

    /* TODO: move this to gpu_utils module */
    k_empty_test<<<1, 512>>>();
    CU_LAUNCH_ERR_SYNC("test kernel");
}

/*! Initilizes neighbor list on the GPU, called at every neighbor search step. 
 */
void init_cudata_nblist(cu_nonbonded_t cu_nb, 
                        const gmx_nblist_t *h_nblist,
                        gmx_bool nonLocal,
                        gmx_bool doStream)
{
    char        sbuf[STRLEN];

    hipStream_t stream     = nonLocal ? cu_nb->timers->nbstream_nl : cu_nb->timers->nbstream;
    cu_nblist_t *d_nblist   = nonLocal ? cu_nb->nblist_nl : cu_nb->nblist;
    //cu_timers_t *timers   = cu_nb->timers;  // FIXME

    if (d_nblist->naps < 0)
    {
        d_nblist->naps = h_nblist->naps;
    }
    else
    {
        if (d_nblist->naps != h_nblist->naps)
        {
            sprintf(sbuf, "In init_cudata_nblist: the #atoms per cell has changed (from %d to %d)",
                    d_nblist->naps, h_nblist->naps);            
            gmx_incons(sbuf);
        }
    }

    realloc_cudata_array((void **)&d_nblist->ci, h_nblist->ci, sizeof(*(d_nblist->ci)),
                         &d_nblist->nci, &d_nblist->ci_nalloc,
                         h_nblist->nci,
                         stream, doStream);

    realloc_cudata_array((void **)&d_nblist->sj4, h_nblist->sj4, sizeof(*(d_nblist->sj4)),
                         &d_nblist->nsj4, &d_nblist->sj4_nalloc,
                         h_nblist->nsj4,
                         stream, doStream);

    realloc_cudata_array((void **)&d_nblist->excl, h_nblist->excl, sizeof(*(d_nblist->excl)),
                         &d_nblist->nexcl, &d_nblist->excl_nalloc,
                         h_nblist->nexcl, 
                         stream, doStream);

    /* need to prune the neighbor list during the next step */
    d_nblist->prune_nbl = TRUE;
}

void cu_move_shift_vec(cu_nonbonded_t cu_nb, 
                       const gmx_nb_atomdata_t *nbatom)
{
    cu_atomdata_t   *adat = cu_nb->atomdata;

    /* HtoD shift vec if we have a dynamic box */
    if (nbatom->dynamic_box || !adat->shift_vec_copied)
    {
        upload_cudata_async(adat->shift_vec, nbatom->shift_vec, SHIFTS * sizeof(*adat->shift_vec), 0);
        adat->shift_vec_copied = TRUE;
    }
}

/* FIXME put all the clear ops into a stream, otherwise it won't overlap with anything  */
void cu_clear_nb_outputs(cu_nonbonded_t cu_nb, 
                         const gmx_nb_atomdata_t *nbatom, // FIXME VEEERY dirty
                         int flags)
{
    hipError_t stat;

    cu_atomdata_t   *adat = cu_nb->atomdata;
    cu_timers_t     *timers = cu_nb->timers;

    gmx_bool calc_ene   = flags & GMX_FORCE_VIRIAL;
    gmx_bool calc_fshift = flags & GMX_FORCE_VIRIAL;

    /* FIXME: this is not a clear OP! */
    cu_move_shift_vec(cu_nb, nbatom);

    stat = hipEventRecord(timers->start_clear, 0);
    CU_RET_ERR(stat, "hipEventRecord on start_clear falied");

    hipMemsetAsync(adat->f, 0, adat->natoms * sizeof(*adat->f), 0);

    /* set the shift force output to 0 */
    if (calc_fshift)
    {
        hipMemsetAsync(adat->f_shift, 0, SHIFTS * sizeof(*adat->f_shift), 0);
    }

    /* set energy outputs to 0 */
    if (calc_ene)
    {
        hipMemsetAsync(adat->e_lj, 0, sizeof(*adat->e_lj), 0);
        hipMemsetAsync(adat->e_el, 0, sizeof(*adat->e_el), 0);
    }

    hipEventRecord(timers->stop_clear, 0);
    CU_RET_ERR(stat, "hipEventRecord on stop_clear falied");

    /* block all future streams until this finishes */
    // XXX this is too restrictive stat = hipStreamWaitEvent(NULL, timers->stop_clear 0);
}

/*! Initilizes atom-data on the GPU, called at every neighbor search step. 
 */
void init_cudata_atoms(cu_nonbonded_t cu_nb,
                       const gmx_nb_atomdata_t *nbat,
                       gmx_bool doStream)
{
    hipError_t stat;
    int         nalloc;
    int         natoms  = nbat->natoms;

    cu_atomdata_t *d_atomd  = cu_nb->atomdata;
    cu_timers_t *timers     = cu_nb->timers;  // FIXME

    /* time async copy */
    stat = hipEventRecord(timers->start_atdat, 0);
    CU_RET_ERR(stat, "hipEventRecord failed on timers->start_atdat");

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven't initilzed yet, i.e d_atomd->natoms == -1 */
    if (natoms > d_atomd->nalloc)
    {
        nalloc = natoms * 1.2 + 100;
    
        /* free up first if the arrays have already been initialized */
        if (d_atomd->nalloc != -1)
        {
            destroy_cudata_array(d_atomd->f, &d_atomd->natoms, &d_atomd->nalloc);
            destroy_cudata_array(d_atomd->xq);
            destroy_cudata_array(d_atomd->atom_types, &d_atomd->ntypes);             
        }
        
        stat = hipMalloc((void **)&d_atomd->f, nalloc*sizeof(*(d_atomd->f)));
        CU_RET_ERR(stat, "hipMalloc failed on d_atomd->f");                   
        stat = hipMalloc((void **)&d_atomd->xq, nalloc*sizeof(*(d_atomd->xq)));
        CU_RET_ERR(stat, "hipMalloc failed on d_atomd->xq");     

        stat = hipMalloc((void **)&d_atomd->atom_types, nalloc*sizeof(*(d_atomd->atom_types)));
        CU_RET_ERR(stat, "hipMalloc failed on d_atomd->atom_types"); 

        d_atomd->nalloc = nalloc;
    }
    
    d_atomd->natoms = natoms;
    d_atomd->natoms_local = nbat->natoms_local;

    if(doStream)
    {
        upload_cudata_async(d_atomd->atom_types, nbat->type, 
                            natoms*sizeof(*d_atomd->atom_types), 0);
    }
    else 
    {
        upload_cudata(d_atomd->atom_types, nbat->type, 
                      natoms*sizeof(*(d_atomd->atom_types)));
    
    }

    stat = hipEventRecord(timers->stop_atdat, 0);
    CU_RET_ERR(stat, "hipEventRecord failed on timers->stop_atdat");

    cu_nb->timings->atomdt_count++;
}

/*! Frees up all GPU resources used for the nonbonded calculations. */
void destroy_cudata(FILE *fplog, cu_nonbonded_t cu_nb)
{
    hipError_t stat;
    cu_atomdata_t       *atomdata;
    cu_nb_params_t      *nb_params;
    cu_nblist_t         *nblist, *nblist_nl;
    cu_timers_t         *timers;

    atomdata    = cu_nb->atomdata;
    nb_params   = cu_nb->nb_params;
    nblist      = cu_nb->nblist;
    nblist_nl   = cu_nb->nblist_nl;
    timers      = cu_nb->timers;

    if (cu_nb == NULL) return;

    if (nb_params->eeltype == cu_eelEWALD)
    {
        cu_unbind_texture("tex_coulomb_tab");
        destroy_cudata_array(nb_params->coulomb_tab, &nb_params->coulomb_tab_size);            
    }

    stat = hipEventDestroy(timers->start_nb);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb");
    stat = hipEventDestroy(timers->stop_nb);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb");
    stat = hipEventDestroy(timers->start_nb_nl);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_nl");
    stat = hipEventDestroy(timers->stop_nb_nl);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_nl");

    stat = hipEventDestroy(timers->start_clear);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_clear");
    stat = hipEventDestroy(timers->stop_clear);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_clear");

    stat = hipEventDestroy(timers->start_atdat);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_atdat");
    stat = hipEventDestroy(timers->stop_atdat);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_atdat");
    stat = hipEventDestroy(timers->start_atdat_nl);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_atdat_nl");
    stat = hipEventDestroy(timers->stop_atdat_nl);
    CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_atdat_nl");

    stat = hipStreamDestroy(timers->nbstream);
    CU_RET_ERR(stat, "hipStreamDestroy failed on nbstream");
    stat = hipStreamDestroy(timers->nbstream_nl);
    CU_RET_ERR(stat, "hipStreamDestroy failed on nbstream_nl");

    if (timers->time_transfers)
    {
        stat = hipEventDestroy(timers->start_nb_h2d);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_h2d");
        stat = hipEventDestroy(timers->stop_nb_h2d);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_h2d");
        stat = hipEventDestroy(timers->start_nb_h2d_nl);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_h2d_nl");
        stat = hipEventDestroy(timers->stop_nb_h2d_nl);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_h2d_nl");

        stat = hipEventDestroy(timers->start_nb_d2h);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_d2h");
        stat = hipEventDestroy(timers->stop_nb_d2h);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_d2h");
        stat = hipEventDestroy(timers->start_nb_d2h_nl);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->start_nb_d2h_nl");
        stat = hipEventDestroy(timers->stop_nb_d2h_nl);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->stop_nb_d2h_nl");
    }

    cu_unbind_texture("tex_nbfp");
    destroy_cudata_array(nb_params->nbfp);

    stat = hipFree(atomdata->shift_vec);
    CU_RET_ERR(stat, "hipEventDestroy failed on atomdata->shift_vec");
    stat = hipFree(atomdata->f_shift);
    CU_RET_ERR(stat, "hipEventDestroy failed on atomdata->f_shift");

    stat = hipFree(atomdata->e_lj);
    CU_RET_ERR(stat, "hipEventDestroy failed on atomdata->e_lj");
    stat = hipFree(atomdata->e_el);
    CU_RET_ERR(stat, "hipEventDestroy failed on atomdata->e_el");

    destroy_cudata_array(atomdata->f, &atomdata->natoms, &atomdata->nalloc);
    destroy_cudata_array(atomdata->xq);
    destroy_cudata_array(atomdata->atom_types, &atomdata->ntypes);            

    destroy_cudata_array(nblist->ci, &nblist->nci, &nblist->ci_nalloc);
    destroy_cudata_array(nblist->sj4, &nblist->nsj4, &nblist->sj4_nalloc);
    destroy_cudata_array(nblist->excl, &nblist->nexcl, &nblist->excl_nalloc);
    destroy_cudata_array(nblist_nl->ci, &nblist_nl->nci, &nblist_nl->ci_nalloc);
    destroy_cudata_array(nblist_nl->sj4, &nblist_nl->nsj4, &nblist_nl->sj4_nalloc);
    destroy_cudata_array(nblist_nl->excl, &nblist_nl->nexcl, &nblist->excl_nalloc);

    stat = hipDeviceReset();
    CU_RET_ERR(stat, "hipDeviceReset failed");

    if (debug)
    {
        fprintf(debug, "Cleaned up CUDA data structures.\n");
    }
}

/*! Frees the device memory pointed by d_ptr and resets the associated 
 *  size and allocation size variables to -1.
 */
static void destroy_cudata_array(void *d_ptr, int *n, int *nalloc)
{
    hipError_t stat;
    
    if (d_ptr)
    {
        stat = hipFree(d_ptr);
        CU_RET_ERR(stat, "hipFree failed");
    }

    if (n)
    {        
        *n = -1;
    }

    if (nalloc)
    {
        *nalloc = -1;
    }
}

/*! Reallocates the device memory pointed by d_ptr and copies the data from the 
 * location pointed by h_src host-side pointer. Allocation is buffered and 
 * therefor freeing is only needed if the previously allocated space is not 
 * enough. 
 */
static void realloc_cudata_array(void **d_dest, void *h_src, size_t type_size, 
                                 int *curr_size, int *curr_alloc_size, 
                                 int req_size, 
                                 hipStream_t stream, gmx_bool doStream)
{
    hipError_t stat;

    if (d_dest == NULL || req_size <= 0)
    {
        return;
    }

    /* reallocate only if the data does not fit = allocation size is smaller 
       than the current requested size */
    if (req_size > *curr_alloc_size)
    {
        /* only free if the array has already been initialized */
        if (*curr_alloc_size >= 0)
        {
            destroy_cudata_array(*d_dest, curr_size, curr_alloc_size);
        }

        *curr_alloc_size = 1.2 * req_size + 100;  /* TODO replace this with a fn pointer 
                                                     passed from outside */

        stat = hipMalloc(d_dest, *curr_alloc_size * type_size);
        CU_RET_ERR(stat, "hipMalloc failed in realloc_cudata_array");
    }

    /* size could have changed without actual reallocation */
    *curr_size = req_size;

    /* upload to device */
    if (h_src)
    {
        if(doStream)
        {
            upload_cudata_async(*d_dest, h_src, *curr_size * type_size, stream);
        }
        else 
        {
            upload_cudata(*d_dest, h_src,  *curr_size * type_size);
        }
    }
}

void cu_move_xq(cu_nonbonded_t cu_nb, const gmx_nb_atomdata_t *nbat,
                gmx_bool nonLocal)
{
    cu_atomdata_t   *d_nbat = cu_nb->atomdata;
    hipStream_t    stream = nonLocal ? cu_nb->timers->nbstream_nl :
                                        cu_nb->timers->nbstream;

    upload_cudata_async(d_nbat->xq, nbat->x,
                        d_nbat->natoms * sizeof(*d_nbat->xq), stream);
}

/*! Blocking waits until the atom data gets copied to the GPU and times the transfer.
 */
void cu_blockwait_atomdata(cu_nonbonded_t cu_nb)
{
    float t;
    cu_blockwait_event(cu_nb->timers->stop_atdat, cu_nb->timers->start_atdat, &t);
    cu_nb->timings->atomdt_h2d_total_time += t;
}

/*! Calculated the ellapsed time during atomdata transfer.
 */
void cu_time_atomdata(cu_nonbonded_t cu_nb)
{
    float t;
    hipError_t stat;

    stat = hipEventElapsedTime(&t, cu_nb->timers->start_atdat, cu_nb->timers->stop_atdat);
    CU_RET_ERR(stat, "hipEventElapsedTime failed in cu_blockwait_nb");
    cu_nb->timings->atomdt_h2d_total_time += t;
}

/*! Synchronizes the respective stream with the atomdata init operation.
 */
void cu_synchstream_atomdata(cu_nonbonded_t cu_nb, gmx_bool nonLocal)
{
    hipError_t stat;
    hipStream_t stream = nonLocal ? cu_nb->timers->nbstream_nl : cu_nb->timers->nbstream;

    stat = hipStreamWaitEvent(stream, cu_nb->timers->stop_atdat, 0);
    CU_RET_ERR(stat, "hipStreamWaitEvent failed");
}

/*! Returns the GPU timing structure or NULL if cu_nb is NULL. */
cu_timings_t * get_gpu_timings(cu_nonbonded_t cu_nb)
{
    return cu_nb != NULL ? cu_nb->timings : NULL;
}

/*! Resets GPU timers. */
void reset_gpu_timings(cu_nonbonded_t cu_nb)
{
    init_timings(cu_nb->timings);
}

/*** Old stuff ***/
int cu_upload_X(cu_nonbonded_t cu_nb, real *h_x) 
{
    cu_atomdata_t *ad = cu_nb->atomdata;

    return upload_cudata(ad->xq, h_x, ad->natoms*sizeof(*ad->xq));
}

int cu_download_F(real *h_f, cu_nonbonded_t cu_nb)
{
    cu_atomdata_t *ad = cu_nb->atomdata;

    return download_cudata(h_f, ad->f, ad->natoms*sizeof(*ad->f));
}
