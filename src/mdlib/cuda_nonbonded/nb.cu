#include "stdlib.h"

#include "smalloc.h"

#include "types/simple.h" 
#include "types/nblist_box.h"
#include "cutypedefs.h"
#include "cudautils.h"

#include "cuda_nb.h"
#include "cuda_data_mgmt.h"
#include "cupmalloc.h"

#define CELL_SIZE               (GPU_NS_CELL_SIZE)
#define CELL_SIZE_POW2_EXPONENT (3) /* NOTE: change this together with GPU_NS_CELL_SIZE !*/
#define NB_DEFAULT_THREADS      (CELL_SIZE * CELL_SIZE)

#include "cutype_utils.cuh"
#include "nb_kernel_utils.cuh"

/* Generate all combinations of force and energy-calculation and/or pruning kernels. */
/** Force only kernels **/
#include "nb_kernels.cuh"
/** Force & energy kernels **/
#define CALC_ENERGIES
#include "nb_kernels.cuh"
#undef CALC_ENERGIES

/*** Neighborlist pruning kernels ***/
/** Force only kernels **/
#define PRUNE_NBL
#include "nb_kernels.cuh"
/** Force & energy kernels **/
#define CALC_ENERGIES
#include "nb_kernels.cuh"
#undef CALC_ENERGIES
#undef PRUNE_NBL

/*! nonbonded kernel function pointer type */
typedef void (*p_k_calc_nb) (const cu_atomdata_t,
                        const cu_nb_params_t, 
                        const cu_nblist_t);

/* XXX
    if GMX_GPU_ENE env var set it always runs the energy kernel unless the 
    GMX_GPU_NO_ENE env var is set, case in which it never runs the energy kernel.     
    --> only for benchmarking purposes */
static gmx_bool alwaysE = (getenv("GMX_GPU_ALWAYS_ENE") != NULL); 
static gmx_bool neverE  = (getenv("GMX_GPU_NEVER_ENE") != NULL);

/*! Returns the number of blocks to be used  for the nonbonded GPU kernel. */
static inline int calc_nb_blocknr(int nwork_units)
{
    int retval = (nwork_units <= GRID_MAX_DIM ? nwork_units : GRID_MAX_DIM);
    if (retval != nwork_units)
    {
        gmx_fatal(FARGS, "Watch out, the number of nonbonded work units exceeds the maximum grid size (%d > %d)!",
                nwork_units, GRID_MAX_DIM);
    }
    return retval;
}

/*! Selects the kernel version (force / energy / pruning) to execute and 
 * returns a function pointer to it. 
 */
static inline p_k_calc_nb select_nb_kernel(int eeltype, gmx_bool doEne, 
                                           gmx_bool doPrune, gmx_bool doKernel2)
{
    p_k_calc_nb k = NULL;

    /* select which kernel will be used */
    switch (eeltype)
    {
        case cu_eelCUT:
            if (!doKernel2)
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_cutoff_forces_1 : 
                                   k_calc_nb_cutoff_forces_prunenbl_1;                                  
                }
                else 
                {
                    k = !doPrune ? k_calc_nb_cutoff_forces_energies_1 :
                                   k_calc_nb_cutoff_forces_energies_prunenbl_1;
                }
            }
            else 
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_cutoff_forces_2 :
                                   k_calc_nb_cutoff_forces_prunenbl_2;
                }
                else 
                {
                    k = !doPrune ? k_calc_nb_cutoff_forces_energies_2 :
                                   k_calc_nb_cutoff_forces_energies_prunenbl_2;
                }
            }
            break;

        case cu_eelRF:
            if (!doKernel2)
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_RF_forces_1 :
                                   k_calc_nb_RF_forces_prunenbl_1;
                }
                else
                {
                    k = !doPrune ? k_calc_nb_RF_forces_energies_1 :
                                   k_calc_nb_RF_forces_energies_prunenbl_1;
                }
            }
            else 
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_RF_forces_2 :
                                   k_calc_nb_RF_forces_prunenbl_2;
                }
                else
                {
                    k = !doPrune ? k_calc_nb_RF_forces_energies_2 :
                                   k_calc_nb_RF_forces_energies_prunenbl_2;
                }
            }
            break;

        case cu_eelEWALD:
            if (!doKernel2)
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_ewald_forces_1 :
                                   k_calc_nb_ewald_forces_prunenbl_1;
                }
                else
                {
                    k = !doPrune ? k_calc_nb_ewald_forces_energies_1:
                                   k_calc_nb_ewald_forces_energies_prunenbl_1;
                }
            }
            else 
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_ewald_forces_2 :
                                   k_calc_nb_ewald_forces_prunenbl_2;
                }
                else
                {
                    k = !doPrune ? k_calc_nb_ewald_forces_energies_2 :
                                   k_calc_nb_ewald_forces_energies_prunenbl_2;
                }
            }
            break;

        default: 
            gmx_incons("The provided electrostatics type does not exist in the  CUDA implementation!");
    }    
    return k;
}

/*!  Launch asynchronously the nonbonded force calculations. 

    This consists of the following (async) steps launched in the default stream 0: 
   - initilize to zero force output;
   - upload x and q;
   - upload shift vector;
   - launch kernel;
   - download forces/energies.
    
    Timing is done using:
    - start_nb/stop_nb events for total execution time;
    - start_nb_h2d/stop_nb_h2d and start_nb_h2d/stop_nb_h2d event for 
    the CPU->GPU and GPU->CPU transfers, respectively.
 */
void cu_stream_nb(cu_nonbonded_t cu_nb,
                  const gmx_nb_atomdata_t *nbatom,                                    
                  gmx_bool calc_ene,
                  gmx_bool sync)
{
    cu_atomdata_t   *atomdata = cu_nb->atomdata;
    cu_nb_params_t  *nb_params = cu_nb->nb_params;
    cu_nblist_t     *nblist = cu_nb->nblist;
    cu_timers_t     *timers = cu_nb->timers;

    int     shmem; 
    int     nb_blocks = calc_nb_blocknr(nblist->nci);
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 
    gmx_bool time_trans = timers->time_transfers; 

    p_k_calc_nb nb_kernel = NULL; /* fn pointer to the nonbonded kernel */

    static gmx_bool doKernel2 = (getenv("GMX_NB_K2") != NULL);        
    static gmx_bool doAlwaysNsPrune = (getenv("GMX_GPU_ALWAYS_NS_PRUNE") != NULL);

    /* XXX debugging code, remove it */
    calc_ene = (calc_ene || alwaysE) && !neverE; 

    if (debug)
    {
        fprintf(debug, "GPU launch configuration:\n\tThread block: %dx%dx%d\n\tGrid: %dx%d\n\t#Cells/Subcells: %d/%d (%d)\n",         
        dim_block.x, dim_block.y, dim_block.z, dim_grid.x, dim_grid.y, nblist->nci*NSUBCELL, 
        NSUBCELL, nblist->naps);
    }
    
    /* beginning of timed nonbonded calculation section */
    hipEventRecord(timers->start_nb, 0);

    /* beginning of timed HtoD section */
    if (time_trans)
    {
        hipEventRecord(timers->start_nb_h2d, 0);
    }

    /* 0 the force output array */
    hipMemsetAsync(atomdata->f, 0, atomdata->natoms * sizeof(*atomdata->f), 0);

    /* HtoD x, q */    
    upload_cudata_async(atomdata->xq, nbatom->x, atomdata->natoms * sizeof(*atomdata->xq), 0);

    /* HtoD shift vec if we have a dynamic box */
    if (nbatom->dynamic_box || !atomdata->shift_vec_copied)
    {
        upload_cudata_async(atomdata->shift_vec, nbatom->shift_vec, SHIFTS * sizeof(*atomdata->shift_vec), 0);   
        atomdata->shift_vec_copied = TRUE;
    }
    
    if (time_trans)
    {
        hipEventRecord(timers->stop_nb_h2d, 0);
    }

    /* set energy outputs to 0 */
    if (calc_ene)
    {
        hipMemsetAsync(atomdata->e_lj, 0.0f, sizeof(*atomdata->e_lj), 0);
        hipMemsetAsync(atomdata->e_el, 0.0f, sizeof(*atomdata->e_el), 0);
    }

    /* launch async nonbonded calculations */        
    /* size of force buffers in shmem */
     shmem = !doKernel2 ?
                (1 + NSUBCELL) * CELL_SIZE * CELL_SIZE * 3 * sizeof(float) :
                CELL_SIZE * CELL_SIZE * 3 * sizeof(float);
     
    nb_kernel = select_nb_kernel(nb_params->eeltype, calc_ene, 
                                 nblist->prune_nbl || doAlwaysNsPrune, doKernel2);
    nb_kernel<<<dim_grid, dim_block, shmem, 0>>>(*atomdata, *nb_params, *nblist);

    if (sync)
    {
        CU_LAUNCH_ERR_SYNC("k_calc_nb");
    }
    else
    {
        CU_LAUNCH_ERR("k_calc_nb");
    }
   
    /* beginning of timed D2H section */
    if (time_trans)
    {
        hipEventRecord(timers->start_nb_d2h, 0);
    }

    /* DtoH f */
    download_cudata_async(nbatom->f, atomdata->f, atomdata->natoms*sizeof(*atomdata->f), 0);
    /* DtoH energies */
    if (calc_ene)
    {
        download_cudata_async(cu_nb->tmpdata.e_lj, atomdata->e_lj, sizeof(*cu_nb->tmpdata.e_lj), 0);
        download_cudata_async(cu_nb->tmpdata.e_el, atomdata->e_el, sizeof(*cu_nb->tmpdata.e_el), 0);
    }

    if (time_trans)
    {        
        hipEventRecord(timers->stop_nb_d2h, 0);
    }

    hipEventRecord(timers->stop_nb, 0);
}


/*! Blocking wait for the asynchrounously launched nonbonded calculations to finish. */
void cu_blockwait_nb(cu_nonbonded_t cu_nb, gmx_bool calc_ene, 
                     float *e_lj, float *e_el)
{    
    hipError_t s;
    float t_tot, t;
    cu_timers_t *timers     = cu_nb->timers;
    cu_timings_t *timings   = cu_nb->timings;

    cu_blockwait_event(timers->stop_nb, timers->start_nb, &t_tot);
    timings->nb_count++;
    
    if (timers->time_transfers)
    {        
        s = hipEventElapsedTime(&t, timers->start_nb_h2d, timers->stop_nb_h2d);
        CU_RET_ERR(s, "hipEventElapsedTime failed in cu_blockwait_nb");
        timings->nb_h2d_time += t;
        t_tot -= t;
        
        s = hipEventElapsedTime(&t, timers->start_nb_d2h, timers->stop_nb_d2h);
        CU_RET_ERR(s, "hipEventElapsedTime failed in cu_blockwait_nb");    
        timings->nb_d2h_time += t;
        t_tot -= t;
    }

    timings->k_time[cu_nb->nblist->prune_nbl ? 1 : 0][calc_ene ? 1 : 0].t += t_tot;
    timings->k_time[cu_nb->nblist->prune_nbl ? 1 : 0][calc_ene ? 1 : 0].c += 1;
   
    /* turn off neighborlist pruning */
    cu_nb->nblist->prune_nbl = FALSE;

    /* XXX debugging code, remove this */
    calc_ene = (calc_ene || alwaysE) && !neverE; 

    if (calc_ene)
    {
        *e_lj += *cu_nb->tmpdata.e_lj;
        *e_el += *cu_nb->tmpdata.e_el;
    }
}

/*! Checks if the nonbonded calculation has finished. */
gmx_bool cu_checkstat_nb(cu_nonbonded_t cu_nb, float *time)
{
    hipError_t stat; 
    cu_timers_t *timers = cu_nb->timers;

    time = NULL;
    stat = hipEventQuery(timers->stop_nb);

    /* we're done, let's calculate times*/
    if (stat == hipSuccess)
    {
        stat = hipEventElapsedTime(time, timers->start_nb, timers->stop_nb);
        CU_RET_ERR(stat, "hipEventElapsedTime on start_nb and stop_nb failed");
    }
    else 
    {
        /* do we have an error? */
        if (stat != hipErrorNotReady) 
        {
            CU_RET_ERR(stat, "the execution of the nonbonded calculations has failed");
        }
    }
    
    return (stat == hipSuccess ? TRUE : FALSE);
}
