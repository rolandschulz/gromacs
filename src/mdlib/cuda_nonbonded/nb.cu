#include "stdlib.h"

#include "smalloc.h"
#include "force.h"
#include "types/simple.h" 
#include "types/nblist_box.h"

#include "cutypedefs.h"
#include "cudautils.h"
#include "cuda_nb.h"
#include "cuda_data_mgmt.h"
#include "cupmalloc.h"

#define CELL_SIZE               (GPU_NS_CELL_SIZE)
#define CELL_SIZE_POW2_EXPONENT (3) /* NOTE: change this together with GPU_NS_CELL_SIZE !*/
#define NB_DEFAULT_THREADS      (CELL_SIZE * CELL_SIZE)

#include "cutype_utils.cuh"
#include "nb_kernel_utils.cuh"

/* Generate all combinations of force and energy-calculation and/or pruning kernels. */
/** Force only kernels **/
#include "nb_kernels.cuh"
/** Force & energy kernels **/
#define CALC_ENERGIES
#include "nb_kernels.cuh"
#undef CALC_ENERGIES

/*** Neighborlist pruning kernels ***/
/** Force only kernels **/
#define PRUNE_NBL
#include "nb_kernels.cuh"
/** Force & energy kernels **/
#define CALC_ENERGIES
#include "nb_kernels.cuh"
#undef CALC_ENERGIES
#undef PRUNE_NBL

/*! nonbonded kernel function pointer type */
typedef void (*p_k_calc_nb)(const cu_atomdata_t,
                            const cu_nb_params_t, 
                            const cu_nblist_t,
                            gmx_bool /*calc virial*/);

/* XXX
    if GMX_GPU_ENE env var set it always runs the energy kernel unless the 
    GMX_GPU_NO_ENE env var is set, case in which it never runs the energy kernel.     
    --> only for benchmarking purposes */
static gmx_bool alwaysE = (getenv("GMX_GPU_ALWAYS_ENE") != NULL); 
static gmx_bool neverE  = (getenv("GMX_GPU_NEVER_ENE") != NULL);

/*! Returns the number of blocks to be used  for the nonbonded GPU kernel. */
static inline int calc_nb_blocknr(int nwork_units)
{
    int retval = (nwork_units <= GRID_MAX_DIM ? nwork_units : GRID_MAX_DIM);
    if (retval != nwork_units)
    {
        gmx_fatal(FARGS, "Watch out, the number of nonbonded work units exceeds the maximum grid size (%d > %d)!",
                nwork_units, GRID_MAX_DIM);
    }
    return retval;
}

/*! Selects the kernel version (force / energy / pruning) to execute and 
 * returns a function pointer to it. 
 */
static inline p_k_calc_nb select_nb_kernel(int eeltype, gmx_bool doEne, 
                                           gmx_bool doPrune, gmx_bool doKernel2)
{
    p_k_calc_nb k = NULL;

    /* select which kernel will be used */
    switch (eeltype)
    {
        case cu_eelCUT:
            if (!doKernel2)
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_cutoff_forces_1 : 
                                   k_calc_nb_cutoff_forces_prunenbl_1;                                  
                }
                else 
                {
                    k = !doPrune ? k_calc_nb_cutoff_forces_energies_1 :
                                   k_calc_nb_cutoff_forces_energies_prunenbl_1;
                }
            }
            else 
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_cutoff_forces_2 :
                                   k_calc_nb_cutoff_forces_prunenbl_2;
                }
                else 
                {
                    k = !doPrune ? k_calc_nb_cutoff_forces_energies_2 :
                                   k_calc_nb_cutoff_forces_energies_prunenbl_2;
                }
            }
            break;

        case cu_eelRF:
            if (!doKernel2)
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_RF_forces_1 :
                                   k_calc_nb_RF_forces_prunenbl_1;
                }
                else
                {
                    k = !doPrune ? k_calc_nb_RF_forces_energies_1 :
                                   k_calc_nb_RF_forces_energies_prunenbl_1;
                }
            }
            else 
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_RF_forces_2 :
                                   k_calc_nb_RF_forces_prunenbl_2;
                }
                else
                {
                    k = !doPrune ? k_calc_nb_RF_forces_energies_2 :
                                   k_calc_nb_RF_forces_energies_prunenbl_2;
                }
            }
            break;

        case cu_eelEWALD:
            if (!doKernel2)
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_ewald_forces_1 :
                                   k_calc_nb_ewald_forces_prunenbl_1;
                }
                else
                {
                    k = !doPrune ? k_calc_nb_ewald_forces_energies_1:
                                   k_calc_nb_ewald_forces_energies_prunenbl_1;
                }
            }
            else 
            {
                if (!doEne)
                {
                    k = !doPrune ? k_calc_nb_ewald_forces_2 :
                                   k_calc_nb_ewald_forces_prunenbl_2;
                }
                else
                {
                    k = !doPrune ? k_calc_nb_ewald_forces_energies_2 :
                                   k_calc_nb_ewald_forces_energies_prunenbl_2;
                }
            }
            break;

        default: 
            gmx_incons("The provided electrostatics type does not exist in the  CUDA implementation!");
    }
    return k;
}

/*!  Launch asynchronously the nonbonded force calculations. 

    This consists of the following (async) steps launched in the default stream 0: 
   - initilize to zero force output;
   - upload x and q;
   - upload shift vector;
   - launch kernel;
   - download forces/energies.
    
    Timing is done using:
    - start_nb/stop_nb events for total execution time;
    - start_nb_h2d/stop_nb_h2d and start_nb_h2d/stop_nb_h2d event for 
    the CPU->GPU and GPU->CPU transfers, respectively.
 */
void cu_stream_nb(cu_nonbonded_t cu_nb,
                  const gmx_nb_atomdata_t *nbatom,                                    
                  // gmx_bool calc_ene,
                  int flags,
                  gmx_bool sync)
{
    cu_atomdata_t   *adat = cu_nb->atomdata;
    cu_nb_params_t  *nb_params = cu_nb->nb_params;
    cu_nblist_t     *nblist = cu_nb->nblist;
    cu_timers_t     *timers = cu_nb->timers;

    int     shmem; 
    int     nb_blocks = calc_nb_blocknr(nblist->nci);
    dim3    dim_block(CELL_SIZE, CELL_SIZE, 1); 
    dim3    dim_grid(nb_blocks, 1, 1); 

    gmx_bool calc_ene   = flags & GMX_FORCE_VIRIAL;
    gmx_bool calc_fshift = flags & GMX_FORCE_VIRIAL;
    gmx_bool time_trans = timers->time_transfers; 

    p_k_calc_nb nb_kernel = NULL; /* fn pointer to the nonbonded kernel */

    static gmx_bool doKernel2 = (getenv("GMX_NB_K2") != NULL);        
    static gmx_bool doAlwaysNsPrune = (getenv("GMX_GPU_ALWAYS_NS_PRUNE") != NULL);

    /* XXX debugging code, remove it */
    calc_ene = (calc_ene || alwaysE) && !neverE; 

    if (debug)
    {
        fprintf(debug, "GPU launch configuration:\n\tThread block: %dx%dx%d\n\tGrid: %dx%d\n\t#Cells/Subcells: %d/%d (%d)\n",         
        dim_block.x, dim_block.y, dim_block.z, dim_grid.x, dim_grid.y, nblist->nci*NSUBCELL, 
        NSUBCELL, nblist->naps);
    }
    
    /* beginning of timed nonbonded calculation section */
    hipEventRecord(timers->start_nb, 0);

    /* beginning of timed HtoD section */
    if (time_trans)
    {
        hipEventRecord(timers->start_nb_h2d, 0);
    }

    /* 0 the force output array */
    hipMemsetAsync(adat->f, 0, adat->natoms * sizeof(*adat->f), 0);

    /* HtoD x, q */    
    upload_cudata_async(adat->xq, nbatom->x, adat->natoms * sizeof(*adat->xq), 0);

    /* HtoD shift vec if we have a dynamic box */
    if (nbatom->dynamic_box || !adat->shift_vec_copied)
    {
        upload_cudata_async(adat->shift_vec, nbatom->shift_vec, SHIFTS * sizeof(*adat->shift_vec), 0);
        adat->shift_vec_copied = TRUE;
    }

    /* set the shift force output to 0 */
    if (calc_fshift)
    {
        hipMemsetAsync(adat->f_shift, 0, SHIFTS * sizeof(*adat->f_shift), 0);
    }

    /* set energy outputs to 0 */
    if (calc_ene)
    {
        hipMemsetAsync(adat->e_lj, 0, sizeof(*adat->e_lj), 0);
        hipMemsetAsync(adat->e_el, 0, sizeof(*adat->e_el), 0);
    }

    if (time_trans)
    {
        hipEventRecord(timers->stop_nb_h2d, 0);
    }

    /* launch async nonbonded calculations */        
    /* size of force buffers in shmem */
     shmem = !doKernel2 ?
                (1 + NSUBCELL) * CELL_SIZE * CELL_SIZE * 3 * sizeof(float) :
                CELL_SIZE * CELL_SIZE * 3 * sizeof(float);
     
    nb_kernel = select_nb_kernel(nb_params->eeltype, calc_ene, 
                                 nblist->prune_nbl || doAlwaysNsPrune, doKernel2);
    nb_kernel<<<dim_grid, dim_block, shmem, 0>>>(*adat, *nb_params, *nblist, 
                                                 calc_fshift);

    if (sync)
    {
        CU_LAUNCH_ERR_SYNC("k_calc_nb");
    }
    else
    {
        CU_LAUNCH_ERR("k_calc_nb");
    }
   
    /* beginning of timed D2H section */
    if (time_trans)
    {
        hipEventRecord(timers->start_nb_d2h, 0);
    }

    /* DtoH f */
    download_cudata_async(nbatom->f, adat->f, adat->natoms*sizeof(*adat->f), 0);

    /* DtoH f_shift */
    if (calc_fshift)
    {
        download_cudata_async(cu_nb->tmpdata.f_shift, adat->f_shift, 
                              SHIFTS * sizeof(*cu_nb->tmpdata.f_shift), 0);
    }

    /* DtoH energies */
    if (calc_ene)
    {
        download_cudata_async(cu_nb->tmpdata.e_lj, adat->e_lj, sizeof(*cu_nb->tmpdata.e_lj), 0);
        download_cudata_async(cu_nb->tmpdata.e_el, adat->e_el, sizeof(*cu_nb->tmpdata.e_el), 0);
    }

    if (time_trans)
    {        
        hipEventRecord(timers->stop_nb_d2h, 0);
    }

    hipEventRecord(timers->stop_nb, 0);
}

/*! Blocking wait for the asynchrounously launched nonbonded calculations to finish. */
void cu_blockwait_nb(cu_nonbonded_t cu_nb, int flags, 
                     float *e_lj, float *e_el, rvec *fshift)
{    
    hipError_t     s;
    int             i;
    float           t_tot, t;
    gmx_bool        calc_ene   = flags & GMX_FORCE_VIRIAL;
    gmx_bool        calc_fshift = flags & GMX_FORCE_VIRIAL;

    cu_timers_t     *timers  = cu_nb->timers;
    cu_timings_t    *timings = cu_nb->timings;
    nb_tmp_data     td = cu_nb->tmpdata;    

    cu_blockwait_event(timers->stop_nb, timers->start_nb, &t_tot);
    timings->nb_count++;
    
    if (timers->time_transfers)
    {        
        s = hipEventElapsedTime(&t, timers->start_nb_h2d, timers->stop_nb_h2d);
        CU_RET_ERR(s, "hipEventElapsedTime failed in cu_blockwait_nb");
        timings->nb_h2d_time += t;
        t_tot -= t;
        
        s = hipEventElapsedTime(&t, timers->start_nb_d2h, timers->stop_nb_d2h);
        CU_RET_ERR(s, "hipEventElapsedTime failed in cu_blockwait_nb");    
        timings->nb_d2h_time += t;
        t_tot -= t;
    }

    timings->k_time[cu_nb->nblist->prune_nbl ? 1 : 0][calc_ene ? 1 : 0].t += t_tot;
    timings->k_time[cu_nb->nblist->prune_nbl ? 1 : 0][calc_ene ? 1 : 0].c += 1;
   
    /* turn off neighborlist pruning */
    cu_nb->nblist->prune_nbl = FALSE;

    /* XXX debugging code, remove this */
    calc_ene = (calc_ene || alwaysE) && !neverE; 

    if (calc_ene)
    {
        *e_lj += *td.e_lj;
        *e_el += *td.e_el;
    }

    if (calc_fshift)
    {
        for (i = 0; i < SHIFTS; i++)
        {
            fshift[i][0] += td.f_shift[i].x;
            fshift[i][1] += td.f_shift[i].y;
            fshift[i][2] += td.f_shift[i].z;
        }
    }
}

/*! Checks if the nonbonded calculation has finished. */
gmx_bool cu_checkstat_nb(cu_nonbonded_t cu_nb, float *time)
{
    hipError_t stat; 
    cu_timers_t *timers = cu_nb->timers;

    time = NULL;
    stat = hipEventQuery(timers->stop_nb);

    /* we're done, let's calculate times*/
    if (stat == hipSuccess)
    {
        stat = hipEventElapsedTime(time, timers->start_nb, timers->stop_nb);
        CU_RET_ERR(stat, "hipEventElapsedTime on start_nb and stop_nb failed");
    }
    else 
    {
        /* do we have an error? */
        if (stat != hipErrorNotReady) 
        {
            CU_RET_ERR(stat, "the execution of the nonbonded calculations has failed");
        }
    }
    
    return (stat == hipSuccess ? TRUE : FALSE);
}
